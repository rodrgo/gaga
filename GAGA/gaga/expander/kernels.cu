
#include <hip/hip_runtime.h>


__global__ void cuda_update_residual(float *d_res, int *d_rows, float *d_updates, int n, int p){
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if (xIndex < n){
		if (abs(d_updates[xIndex]) > 0){
                	int idx = xIndex*p;
			float update = -d_updates[xIndex]; 
			for (int i = 0; i < p; i++){
				atomicAdd(d_res + d_rows[idx + i], update);
			}
		} 
	}
}

// Not tested yet
/*
__global__ void cuda_update_residual_np(float *d_res, int *d_rows, float *d_updates, int n, int p){
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < n*p){
                int idx = (int) (tid/p);
		float update = -d_updates[idx]; 
		if (abs(d_updates[xIndex]) > 0){
			atomicAdd(d_res + d_rows[idx + i], update);
		}
	}
}
*/

/*
*************************************************
* 	For tranforming to row-major order	*
*************************************************
*/

// nonzero_rows_count[i] = number of nonzeros in row i.
__global__ void count_nonzeros_in_rows_index(float *nonzero_rows_count, int *rm_rows_index, int m){
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if ( xIndex < m){
		nonzero_rows_count[xIndex] = (float)(rm_rows_index[2*xIndex + 1] - rm_rows_index[2*xIndex] + 1);
	}
}

	// Create vector 2-m vector [a_1, a_2, b_1, b_2, ..., x_1, x_2] indicating start and end position of columns in row i of matrix
	// i.e. if row_i has only one nonzero, then a_i == b_i
	__global__ void create_row_index(int *rm_rows, int *rm_rows_index, int m, int np){
		unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
		if ( xIndex < np){
			if (xIndex > 0 && xIndex < np - 1){
				if (rm_rows[xIndex - 1] < rm_rows[xIndex]){
					rm_rows_index[2*rm_rows[xIndex]] = xIndex;
				}
				if(rm_rows[xIndex] < rm_rows[xIndex + 1]){
					rm_rows_index[2*rm_rows[xIndex] + 1] = xIndex;
				}
			}
			if (xIndex == 0){
				rm_rows_index[2*rm_rows[xIndex]] = xIndex;
				if(rm_rows[xIndex] < rm_rows[xIndex + 1])
					rm_rows_index[2*rm_rows[xIndex] + 1] = xIndex;
			}
			if (xIndex == np - 1){
				if (rm_rows[xIndex - 1] < rm_rows[xIndex])
					rm_rows_index[2*rm_rows[xIndex]] = xIndex;
				rm_rows_index[2*rm_rows[xIndex] + 1] = xIndex;
			}
		}
	}


	/*
	*************************************************
	*						*
	* 			SMP			*
	*						*
	*************************************************
	*/

	__global__ void cuda_compute_scores_smp(float *d_u, float *d_b, int *d_rows, float *d_aux, int n, int p)
	{
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if(tid < n){
			int idx = p*tid;
			float tmp = 0.0;
			float median = 0.0;

			for(int i = 0; i < p; i++)
				d_aux[idx + i] = (float) d_b[d_rows[idx + i]];
		
			// sort chunk in d_aux with bubblesort. O(p^2) work for a thread, but O(1) memory 
			// We exploit computational power

			for(int i = 0; i < p; i++)
				for(int j = 0; j < p - 1; j++)
					if(d_aux[idx + j] > d_aux[idx + j + 1]){
						tmp = d_aux[idx + j + 1];
						d_aux[idx + j + 1] = d_aux[idx + j];
						d_aux[idx + j] = tmp;
					}	

			median = (p % 2) == 0 ? (d_aux[idx + (p/2) - 1] + d_aux[idx + (p/2)])/2 : d_aux[idx + (p - 1)/2];	
			d_u[tid] = median;
		}
	}


	/*
	*************************************************
	*						*
	* 			SSMP			*
	*						*
	*************************************************
	*/

	__global__ void cuda_compute_scores_ssmp(float *d_scores, float *d_medians, float *d_res, int *d_rows, float *d_vals, float *d_aux, int n, int p)
	{
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if(tid < n){

			int idx = p*tid;
			float tmp = 0.0;
			float z = 0.0;

			for(int i = 0; i < p; i++)
				d_aux[idx + i] = d_res[d_rows[idx + i]];

			// sort chunk in d_aux with bubblesort. O(p^2) work for a thread, but O(1) memory 
			// We exploit computational power

			for(int i = 0; i < p; i++)
				for(int j = 0; j < p - 1; j++)
					if(d_aux[idx + j] > d_aux[idx + j + 1]){
						tmp = d_aux[idx + j + 1];
						d_aux[idx + j + 1] = d_aux[idx + j];
						d_aux[idx + j] = tmp;
					}

			// z = median
			z = (p % 2) == 0 ? (d_aux[idx + (p/2) - 1] + d_aux[idx + (p/2)])/2 : d_aux[idx + (p - 1)/2];

			d_medians[tid] = z;
			d_scores[tid] = abs(z);

		}
	}

	__global__ void cuda_update_scores_ssmp(float *d_scores, float *d_medians, float *d_res, int *d_rows, float *d_aux, int *d_rm_cols, int *d_rm_rows_index, int max_nonzero_rows_count, int p, int h_node)
	{
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if(tid < max_nonzero_rows_count*p){
			int section = (int)(tid/max_nonzero_rows_count); // from 0 to p-1
			int row = d_rows[p*h_node + section];
			int num_nonzeros_in_row = d_rm_rows_index[2*row + 1] - d_rm_rows_index[2*row] + 1; // from 1 to max_nonzero_rows_count
			int max_position = num_nonzeros_in_row - 1; // from 0 to (max_nonzero_rows_count - 1)
			int position_in_section = tid - section*max_nonzero_rows_count; // from 0 to (max_nonzero_rows_count - 1) 
			if (position_in_section <= max_position){

				int node = d_rm_cols[d_rm_rows_index[2*row] + position_in_section];
				int row_idx = p*node;
				int idx = p*tid;
				float tmp = 0.0;
				float z = 0.0;

				for(int i = 0; i < p; i++)
					d_aux[idx + i] = d_res[d_rows[row_idx + i]];

				// sort chunk in d_aux with bubblesort. O(p^2) work for a thread, but O(1) memory 

				for(int i = 0; i < p; i++)
					for(int j = 0; j < p - 1; j++)
						if(d_aux[idx + j] > d_aux[idx + j + 1]){
							tmp = d_aux[idx + j + 1];
							d_aux[idx + j + 1] = d_aux[idx + j];
							d_aux[idx + j] = tmp;
						}

				// z = median
				z = (p % 2) == 0 ? (d_aux[idx + (p/2) - 1] + d_aux[idx + (p/2)])/2 : d_aux[idx + (p - 1)/2];

				d_medians[node] = z;
				d_scores[node] = abs(z);
			}
		}
	}


	/*
	*************************************************
	*						*
	* 			ER			*
	*						*
	*************************************************
	*/

	__global__ void cuda_compute_scores_er(float *d_scores, float *d_modes, float *d_res, int *d_rows, float *d_vals, float *d_aux, int n, int p)
	{
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if(tid < n){

			int idx = p*tid;
			float tmp = 0.0;
			float EPS = 0.0000001;

			for(int i = 0; i < p; i++)
				d_aux[idx + i] = d_res[d_rows[idx + i]];

			// sort chunk in d_aux with bubblesort. O(p^2) work for a thread, but O(1) memory 
			// We exploit computational power

			for(int i = 0; i < p; i++)
				for(int j = 0; j < p - 1; j++)
					if(d_aux[idx + j] > d_aux[idx + j + 1]){
						tmp = d_aux[idx + j + 1];
						d_aux[idx + j + 1] = d_aux[idx + j];
						d_aux[idx + j] = tmp;
					}

			// Compute modes

			float mode = d_aux[idx];
			float count = 1;
			float countMode = 1;

			for (int i = 1; i < p; i++){
				if (abs(d_aux[idx + i] - d_aux[idx + i - 1]) <= EPS){
					count = count + 1;
					if (count > countMode){
						countMode = count;
						mode = d_aux[idx + i - 1];
					}
				}else{
					count = 1;
				}
			}

			d_modes[tid] = mode;
			d_scores[tid] = abs(mode) > EPS ? countMode : 0;

		}
	}

	/*
	0 <= tid < max_nonzero_rows_count*p
	[section_0 | section_1 | ... | section_{p-1}]

	*/
	__global__ void cuda_update_scores_er(float *d_scores, float *d_modes, float *d_res, int *d_rows, float *d_aux, int *d_rm_cols, int *d_rm_rows_index, int max_nonzero_rows_count, int p, int h_node)
	{
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if(tid < max_nonzero_rows_count*p){
			int section = (int)(tid/max_nonzero_rows_count); // from 0 to p-1
			int row = d_rows[p*h_node + section];
			int num_nonzeros_in_row = d_rm_rows_index[2*row + 1] - d_rm_rows_index[2*row] + 1; // from 1 to max_nonzero_rows_count
			int max_position = num_nonzeros_in_row - 1; // from 0 to (max_nonzero_rows_count - 1)
			int position_in_section = tid - section*max_nonzero_rows_count; // from 0 to (max_nonzero_rows_count - 1) 
			if (position_in_section <= max_position){

				int node = d_rm_cols[d_rm_rows_index[2*row] + position_in_section];
				int row_idx = p*node;
				int idx = p*tid;
				float tmp = 0.0;
				float EPS = 0.0000001;

				for(int i = 0; i < p; i++)
					d_aux[idx + i] = d_res[d_rows[row_idx + i]];

				// sort chunk in d_aux with bubblesort. O(p^2) work for a thread, but O(1) memory 

				for(int i = 0; i < p; i++)
					for(int j = 0; j < p - 1; j++)
						if(d_aux[idx + j] > d_aux[idx + j + 1]){
							tmp = d_aux[idx + j + 1];
							d_aux[idx + j + 1] = d_aux[idx + j];
							d_aux[idx + j] = tmp;
						}

				// Compute modes
				float mode = d_aux[idx];
				float count = 1;
				float countMode = 1;

				for (int i = 1; i < p; i++){
					if (abs(d_aux[idx + i] - d_aux[idx + i - 1]) <= EPS){
						count = count + 1;
						if (count > countMode){
							countMode = count;
							mode = d_aux[idx + i - 1];
						}
					}else{
						count = 1;
					}
				}

				d_modes[node] = mode;
				d_scores[node] = abs(mode) > EPS ? countMode : 0;
			}
		}
	}


	/*
	*************************************************
	*						*
	* 		Parallel-LDDSR			*
	*						*
	*************************************************
	*/

	__global__ void cuda_compute_scores_lddsr(float *d_scores, float *d_res, float *d_vec, float *d_updates, int *d_rows, int *d_foundCandidates, int n, int d, int shift)
	{
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		
		if (tid < n){
			float omega = 0.0;
			int idx = d*tid;
			float EPS = 0.000001;
			float thresh = ((float) d)/2;

			d_scores[tid] = 0;
			omega = d_res[d_rows[idx + shift]];
			d_updates[tid] = 0;

			if (abs(omega) > EPS){
				for (int i = 0; i < d; i++){
					if (abs(d_res[d_rows[idx + i]] - omega) <= EPS)
						d_scores[tid]++;
				}
				if (d_scores[tid] > thresh){
					d_foundCandidates[0] = 1;
					d_vec[tid] = d_vec[tid] + omega; 
					/*
					// update residual
					for (int i = 0; i < d; i++){
						atomicAdd(d_res + d_rows[idx + i], -omega);
					}
					*/
					d_updates[tid] = omega;
				}
				//d_updates[tid] = d_vec[tid] + omega; // Use this when using h_found_candidates solution
			}

		}
	}

	/*
	*************************************************
	*						*
	* 		Parallel-L0			*
	*						*
	*************************************************
	*/

	__global__ void cuda_compute_scores_l0(float *d_scores, float *d_res, float *d_vec, float *d_updates, int *d_rows, int *d_foundCandidates, int n, int d, int thresh, int shift)
	{
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		
		if (tid < n){
			float omega = 0.0;
			int idx = d*tid;
			float EPS = 0.000001;
			//float thresh = 1;

			d_scores[tid] = 0;
			omega = d_res[d_rows[idx + shift]];
			d_updates[tid] = 0;

			if (abs(omega) > EPS){
				for (int i = 0; i < d; i++){
					if (abs(d_res[d_rows[idx + i]] - omega) <= EPS)
						d_scores[tid]++;
					if (abs(d_res[d_rows[idx + i]]) <= EPS)
						d_scores[tid]--;
				}
				if (d_scores[tid] > thresh){
					d_foundCandidates[0] = 1;
					d_vec[tid] = d_vec[tid] + omega; 
					/*
					// update residual
					for (int i = 0; i < d; i++){
						atomicAdd(d_res + d_rows[idx + i], -omega);
					}
					*/
					d_updates[tid] = omega;
					//d_updates[tid] = d_vec[tid] + omega;
				}
			}
		}
	}


	/*
	*************************************************
	*						*
	* 		Robust-L0			*
	*						*
	*************************************************
	*/

	__constant__ float sigma_noise; 
	__constant__ float prob_zero_factor;
	__constant__ float prob_equal_factor;
	__constant__ float sigma_signal_zero;
	__constant__ float sigma_signal_equal;

	__device__ int bernoulli(float prob, float unif_rv){
		float value = 0.0;
		if (unif_rv <= prob){
			value = 1.0;
		}
		return value;
	}

	__device__ float norm_pdf_ratio(float x, float sigma_1, float sigma_2){
		return (sigma_2/sigma_1)*expf(x*x/(2*sigma_2*sigma_2) - x*x/(2*sigma_1*sigma_1));
	}

	__device__ float probability_zero(float omega){
		return 1/(1 + prob_zero_factor*norm_pdf_ratio(omega, sigma_signal_zero, sigma_noise));
	}

	__device__ float probability_equal(float omega){
		return 1/(1 + prob_equal_factor*norm_pdf_ratio(omega, sigma_signal_equal, sqrtf(2)*sigma_noise));
	}

	__global__ void cuda_sample_prob_zero_u(int *d_bernoulli_pz_u, float *d_unif, int d, int n, float *d_res, int * d_rows, int shift){
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if (tid < n){
			int unif_idx = (2*d + 1)*tid;
			float unif = d_unif[unif_idx]; 
			float omega = d_res[d_rows[d*tid + shift]];
			float prob_zero_u = probability_zero(omega);
			d_bernoulli_pz_u[tid] = bernoulli(prob_zero_u, unif);
		}
	}

	__global__ void cuda_compute_scores(int *d_scores, int* d_bernoulli_pz_u, float *d_average_updates, float *d_unif, int d, int n, float *d_res, int *d_rows, int shift){
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if (tid < n){
			if (d_bernoulli_pz_u[tid] == 0){ 
				int unif_idx = (2*d + 1)*tid + 1;
				float omega = d_res[d_rows[d*tid + shift]];
				float v;
				float unif;
				float prob_equal;
				float prob_zero;
				int is_equal = 0;
				int is_zero = 0;

				float num_equal = 0.0;
				float sum_updates = 0.0;
				int score = 0;
				for (int i = 0; i < d; i++){
					// get value
					v = d_res[d_rows[d*tid + i]];

					// compute Prob(v = omega)
					prob_equal = probability_equal(v - omega);
					unif = d_unif[unif_idx++]; 
					is_equal = bernoulli(prob_equal, unif);

					// compute Prob(v = 0)
					prob_zero = probability_zero(v);
					unif = d_unif[unif_idx++]; 
					is_zero = bernoulli(prob_zero, unif);

					if (is_equal == 1){
						score += 1;
						sum_updates += v;
						num_equal += 1;
					}
					if (is_zero == 1){
						score -= 1;
					}
				}
				d_scores[tid] = score;
				if (num_equal > 0){
					d_average_updates[tid] = sum_updates/num_equal;
				}else{
					d_average_updates[tid] = 0;
				}
			}else{
				d_average_updates[tid] = 0;
			}

		}
	}

	__global__ void cuda_update_signal_robust_l0(float *d_res, int *d_bernoulli_pz_u, float *d_vec, float *d_updates, float *d_average_updates, int *d_rows, int n, int d, float alpha, int *d_scores, int enforce_l1_decrease){

		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if (tid < n){
			if (d_bernoulli_pz_u[tid] == 0){
				int score = d_scores[tid];
				if (((float) score) >= alpha){

					int idx = d*tid;
					//float omega = 0.0;
					float update = 0.0;

					update = d_average_updates[tid];
					//omega = d_res[d_rows[idx + shift]];

					int do_update = 1;

					if (enforce_l1_decrease == 1){
						float new_energy = 0.0;
						float old_energy = 0.0;
						for (int i = 0; i < d; i++){
							old_energy += abs(d_res[d_rows[idx + i]]);
							new_energy += abs(d_res[d_rows[idx + i]] - update);
						}
						if (new_energy > old_energy){
							do_update = 0;
						}
					}

					
					if (do_update == 1){
						d_vec[tid] = d_vec[tid] + update;
						d_updates[tid] = update;
					}
				}
			}
		}
	}

	/*
	*************************************************
	*						*
	* 	Deterministic Robust-L0			*
	*						*
	*************************************************
	*/

	__global__ void cuda_deterministic_prob_zero_u(float *d_pz_u, int d, int n, float *d_res, int * d_rows, int shift){
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if (tid < n){
			float omega = d_res[d_rows[d*tid + shift]];
			float prob_zero_u = probability_zero(omega);
			d_pz_u[tid] = prob_zero_u;
		}
	}

	__global__ void cuda_compute_scores_det_robust_l0(int *d_scores, float* d_pz_u, float *d_average_updates, int d, int n, float *d_res, int *d_rows, int shift, float prob_thresh){
		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if (tid < n){
			if (d_pz_u[tid] <= 1 - prob_thresh){ 
				float omega = d_res[d_rows[d*tid + shift]];
				float v;

				float num_equal = 0.0;
				float sum_updates = 0.0;
				int score = 0;
				for (int i = 0; i < d; i++){

					// get value
					v = d_res[d_rows[d*tid + i]];

					// compute Prob(v = omega)
					if (probability_equal(v - omega) >= prob_thresh){
						score += 1;
						sum_updates += v;
						num_equal += 1;
					}

					// compute Prob(v = 0)
					if (probability_zero(v) >= 1 - prob_thresh){
						score -= 1;
					}

				}
				d_scores[tid] = score;
				if (num_equal > 0){
					d_average_updates[tid] = sum_updates/num_equal;
				}else{
					d_average_updates[tid] = 0;
				}
			}else{
				d_average_updates[tid] = 0;
			}

		}
	}

	__global__ void cuda_update_signal_det_robust_l0(float *d_res, float *d_vec, float *d_updates, float *d_average_updates, int *d_rows, int n, int d, float alpha, int *d_scores, int enforce_l1_decrease){

		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if (tid < n){
			int score = d_scores[tid];
			if (((float) score) >= alpha){

				int idx = d*tid;
				//float omega = 0.0;
				float update = 0.0;

				update = d_average_updates[tid];
				//omega = d_res[d_rows[idx + shift]];

				int do_update = 1;

				if (enforce_l1_decrease == 1){
					float new_energy = 0.0;
					float old_energy = 0.0;
					for (int i = 0; i < d; i++){
						old_energy += abs(d_res[d_rows[idx + i]]);
						new_energy += abs(d_res[d_rows[idx + i]] - update);
					}
					if (new_energy > old_energy){
						do_update = 0;
					}
				}

				
				if (do_update == 1){
					d_vec[tid] = d_vec[tid] + update;
					d_updates[tid] = update;
				}
			}
		}
	}

	__global__ void cuda_det_robust_l0_step(float *d_res, float *d_vec, int *d_rows, int n, int d, float alpha, int shift, float prob_thresh, int enforce_l1_decrease){

		int tid = threadIdx.x + blockDim.x*blockIdx.x;
		if (tid < n){

			int idx = d*tid;
			float omega = d_res[d_rows[d*tid + shift]];
			float prob_zero_u = probability_zero(omega);
			float update = 0.0;
			int score = 0;

			if (prob_zero_u <= 1 - prob_thresh){ 
				float v;

				float num_equal = 0.0;
				float sum_updates = 0.0;
				for (int i = 0; i < d; i++){

					// get value
					v = d_res[d_rows[d*tid + i]];

					// compute Prob(v = omega)
					if (probability_equal(v - omega) >= prob_thresh){
						score += 1;
						sum_updates += v;
						num_equal += 1;
					}

					// compute Prob(v = 0)
					if (probability_zero(v) >= 1 - prob_thresh){
						score -= 1;
					}

				}
				if (num_equal > 0){
					update = sum_updates/num_equal;
				}else{
					update = 0;
				}
			}else{
				update = 0;
			}

			// update

			if (((float) score) >= alpha){

				//float omega = 0.0;

				//omega = d_res[d_rows[idx + shift]];

				int do_update = 1;

				if (enforce_l1_decrease == 1){
					float new_energy = 0.0;
					float old_energy = 0.0;
					for (int i = 0; i < d; i++){
						old_energy += abs(d_res[d_rows[idx + i]]);
						new_energy += abs(d_res[d_rows[idx + i]] - update);
					}
					if (new_energy > old_energy){
						do_update = 0;
					}
				}

				
				if (do_update == 1){
					d_vec[tid] = d_vec[tid] + update;
				}
			}
		}
	}
	

	/*
	*************************************************
	*
	* 	Adaptive-Robust-L0			
	*						
	*************************************************
	*/
	__constant__ float sigma2_n; 
	__constant__ float sigma2_s; 
	__constant__ float snr; 
	__constant__ int boost_flag;

	__device__ float R2(float z){
		float res = expf(z) - 1;
		res -= z;
		res -= z*z/2;
		return res;
	}

	__device__ float R3(float z){
		float res = expf(z) - 1;
		res -= z;
		res -= z*z/2;
		res -= z*z*z/6;
		return res;
	}

	__device__ float pdf_ratio(float x, float t, float q){
		return sqrtf(t/(q*snr + t))*expf(-(x*x)/(2*sigma2_n)*(1/(q*snr + t) - 1/t));
	}


	__device__ float pdf_ratio_tail(float x, float t, float z){
		float a = z*R2(z)/R3(z);
		return sqrtf(t/(snr*a + 1))*expf(-(x*x)/(2*sigma2_n)*(1/(snr*a + 1) - 1/t));
	}

	__device__ float prob(float t, float x, float d_rho){
		float res = 0.0;
		float z = t*d_rho;
		res = 1;
		res += z*pdf_ratio(x, t, 1.0f);
		res += (z*z/2)*pdf_ratio(x, t, 2.0f);
		res += (z*z*z/6)*pdf_ratio(x, t, 3.0f);
		res += (z*z*z*z/24)*pdf_ratio(x, t, 4.0f);
		//res += R3(z)*pdf_ratio_tail(x, t, z);
		return 1/(1 + res);
	}

	// BEGIN: Probabilities with residual


	__device__ float res_prob_2(float z){
		float res = expf(z);
		res -= 1;
		res -= z;
		res -= z*z/2;
		return res;
	}

	__device__ float res_prob_3(float z){
		float res = expf(z);
		res -= 1;
		res -= z;
		res -= z*z/2;
		res -= (z*z*z)/6;
		return res;
	}

	__device__ float sigma2_tail_wr_3(float t, float d_rho, float r3){
		//return sigma2_n*(snr*t*d_rho*res_prob_2(t, d_rho)/res_prob_3(t, d_rho) + t);
		float z = t*d_rho;
		float r2 = r3 + (z*z*z)/6;
		return ((sigma2_s*t*d_rho)*r2 + t*sigma2_n*r3)/r3;
	}

	__device__ float pdf_ratio_tail_wr_3(float x, float t, float d_rho, float r3){
		float sigma2_tail = sigma2_tail_wr_3(t, d_rho, r3);
		return sqrtf(t*sigma2_n/sigma2_tail)*expf(-(x*x)/2*(1/sigma2_tail - 1/(t*sigma2_n)));
	}

	__device__ float prob_wr(float t, float x, float d_rho, float rho){
		float res = 0.0;
		float z = t*d_rho;
		float factor = 1.0;

		res += 1;
		factor = factor*(z/1);
		res += factor*pdf_ratio(x, t, 1.0f);
		factor = factor*(z/2);
		res += factor*pdf_ratio(x, t, 2.0f);
		factor = factor*(z/3);
		res += factor*pdf_ratio(x, t, 3.0f);

		
		float res_test = res;
		float r3 = res_prob_3(z);
		res_test += r3*pdf_ratio_tail_wr_3(x, t, d_rho, r3);

		/*

		factor = factor*(z/4);
		res += factor*pdf_ratio(x, t, 4.0f);
		factor = factor*(z/5);
		res += factor*pdf_ratio(x, t, 5.0f);
		factor = factor*(z/6);
		res += factor*pdf_ratio(x, t, 6.0f);
		factor = factor*(z/7);
		res += factor*pdf_ratio(x, t, 7.0f);
		factor = factor*(z/8);
		res += factor*pdf_ratio(x, t, 8.0f);

		factor = factor*(z/9);
		res += factor*pdf_ratio(x, t, 9.0f);
		factor = factor*(z/10);
		res += factor*pdf_ratio(x, t, 10.0f);

		*/

		/*
		res_test = res;


		factor = factor*(z/11);
		res += factor*pdf_ratio(x, t, 11.0f);
		factor = factor*(z/12);
		res += factor*pdf_ratio(x, t, 12.0f);
		factor = factor*(z/13);
		res += factor*pdf_ratio(x, t, 13.0f);
		if (abs(1/res - 1/res_test) >=1e-3){
			printf("%g, rho=%g, ERROR=%g\n", t, rho, abs(1/res - 1/res_test));
		}
		*/
		
		return 1/(res);

	}

	__device__ float prob_zero(float x, int d, int k, int m){
		float d_rho = ((float) d)*((float) k)/((float) m);
		float rho = ((float)k)/((float)m);
		float p = prob_wr(1, x, d_rho, rho);
		return p;
		//return prob_wr(1, x, d_rho);
	}

	__device__ float prob_equal(float x, int d, int k, int m){
		float d_rho = ((float) d)*((float) k)/((float) m);
		float rho = ((float)k)/((float)m);
		float p = prob_wr(2, x, d_rho, rho);
		return p;
		//return prob_wr(2, x, d_rho);
	}

	__global__ void compute_prob_zero_at_zero(float *d_prob_zero_at_zero, int d, int k, int m){
		int tid = threadIdx.x + blockDim.x*blockIdx.x;

		if (tid == 0){
			d_prob_zero_at_zero[tid] = prob_zero(0, d, k, m);
		}

	}

	__global__ void get_probs_nonzero(float *d_probs_nonzero, float *d_res, int d, int k, int m){

		int tid = threadIdx.x + blockDim.x*blockIdx.x;

		if (tid < m){
			float omega = d_res[tid];
			d_probs_nonzero[tid] = 1 - prob_zero(omega, d, k, m);
		}

	}

	/*
	__device__ float prob_zero(float x, int d, int k, int m){
		float d_rho = ((float) d)*((float) k)/((float) m);
		return prob(1, x, d_rho);
	}

	__device__ float prob_equal(float x, int d, int k, int m){
		float d_rho = ((float) d)*((float) k)/((float) m);
		return 2*prob(2, x, d_rho);
	}

	__device__ float prob(float t, float x, float d_rho){
		float res = 0.0;
		float z = t*d_rho;
		res = 1;
		res += z*pdf_ratio(x, t, 1.0f);
		res += (z*z/2)*pdf_ratio(x, t, 2.0f);
		res += (z*z*z/6)*pdf_ratio(x, t, 3.0f);
		res += (z*z*z*z/24)*pdf_ratio(x, t, 4.0f);
		return 1/(1 + res);
	}

	*/


	// END: Probabilities with residual


	__global__ void find_nonzeros(float *d_vec, float *d_vec_ind, int n, int d, int k, int m, float prob_thresh, float *d_prob_zero_at_zero){

		int tid = threadIdx.x + blockDim.x*blockIdx.x;

		if (tid < n){
			// Probability of nonzero being large
			d_vec_ind[tid] = prob_zero(d_vec[tid], d, k, m)/d_prob_zero_at_zero[0];

			/*
			if(1 - prob_zero(d_vec[tid], d, k, m) >= prob_thresh)
				d_vec_ind[tid] = 1;
			*/
		}

	}

	__global__ void cuda_adaptive_robust_l0_score_and_update(float * d_vec, int alpha, int d, int k, int m, int n, float *d_res, int *d_rows, int shift, float prob_thresh, float *d_prob_zero_at_zero){

		int tid = threadIdx.x + blockDim.x*blockIdx.x;

		if (tid < n){

			int idx = d*tid;
			float omega = d_res[d_rows[idx + shift]];
			float prob_nonzero = 0.0;
			float score = 0;
			float update = 0.0;

			prob_nonzero = 1 - prob_zero(omega, d, k, m)/d_prob_zero_at_zero[0];

			if (prob_nonzero >= prob_thresh){

				float v;
				float pe = 0.0;
				float pz = 0.0;
				float pe_zero = prob_equal(0, d, k, m);
				float pz_zero = prob_zero(0, d, k, m);
				
				float num_eq = 0;

				if (boost_flag == 1){
					float pe_sum = 0.0;
					float pz_sum = 0.0;
					for (int i = 0; i < d; i++){
						v = d_res[d_rows[idx + i]];
						pe = prob_equal(v - omega, d, k, m);
						pe_sum += pe; 
						pz_sum += prob_zero(v, d, k, m);
						update += pe*v;
						//num_eq += 1;
					}
					score = (pe_sum/pe_zero - pz_sum/pz_zero);
					update = update/pe_sum;
					//update = update/num_eq;
					//update = omega;
				}else{
					for (int i = 0; i < d; i++){
						v = d_res[d_rows[idx + i]];
						pe = prob_equal(v - omega, d, k, m)/pe_zero;
						pz = prob_zero(v, d, k, m)/pz_zero;
						if (pe >= prob_thresh){
							update += v;
							score += 1;
							num_eq += 1;
						}
						if (pz >= 1 - prob_thresh){
							score -= 1;
						}
					}
					if (score < 2)
						score = 0.0f;
					update = update/num_eq;
				}

			}else{

				score = 0.0f;
				update = 0.0f;

			}

			if (score >= ((float) alpha) - 0.05){

				float new_energy = 0.0;
				float old_energy = 0.0;

				for (int i = 0; i < d; i++){
					old_energy += abs(d_res[d_rows[idx + i]]);
					new_energy += abs(d_res[d_rows[idx + i]] - update);
				}
				if (new_energy < old_energy){
					d_vec[tid] = d_vec[tid] + update;
				}
			}
		}
	}


/*
*************************************************
*						*
* For exact thresholding and residual update	*
*						*
*************************************************
*/

__global__ void thresholdK(float *d_vec, int *d_bin, int k_bin, int * offset, int n){
        unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

        if( (xIndex < n) & (d_bin[xIndex] == k_bin) ){
                if(atomicSub(offset, 1) - 1 >= 0){
                        d_vec[xIndex] = 0;
                }
        }
}

__global__ void cudaUpdateSignal(float *d_vec, int h_node, float *d_updates){

        int tid = threadIdx.x + blockDim.x*blockIdx.x;

        if(tid == h_node){
                d_vec[tid] += d_updates[h_node];
        }
}

__global__ void cudaUpdateResidual(float *d_res, int *d_rows, float *d_vals, float *d_medians, int h_node, int p){

        int tid = threadIdx.x + blockDim.x*blockIdx.x;

        if(tid == h_node){
                int idx = tid*p;
                for(int i = 0; i < p; i++)
                        d_res[d_rows[idx + i]] -= d_medians[tid]*(d_vals[idx + i]);
        }
}

__global__ void countSupport( float *d_vec, int n, int *dev_c ) { 
 
        int tid = threadIdx.x + blockIdx.x * blockDim.x; 
        if( tid < n ) { 
                if(d_vec[tid] != 0){ 
                        atomicAdd( dev_c , 1 ); 
                } 
        } 
} 

