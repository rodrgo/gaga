#include "hip/hip_runtime.h"
/* Copyright 2010-2013 Jeffrey D. Blanchard and Jared Tanner
 *   
 * GPU Accelerated Greedy Algorithms for Compressed Sensing
 *
 * Licensed under the GAGA License available at gaga4cs.org and included as GAGA_license.txt.
 *
 * In  order to use the GAGA library, or any of its constituent parts, a user must
 * agree to abide by a set of * conditions of use. The library is available at no cost 
 * for ``Internal'' use. ``Internal'' use of the library * is defined to be use of the 
 * library by a person or institution for academic, educational, or research purposes 
 * under the conditions in the included GAGA_license.txt. Any use of the library implies 
 * that these conditions have been understood, and that the user agrees to abide by all 
 * the listed conditions.
 *     
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * Any redistribution or derivatives of this software must contain this header in all files
 * and include a copy of GAGA_license.txt.
 */

__global__ void lookForValue(float *d_vec, float *d_value, int n){
        unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
        if( (xIndex < n) ){
                if( abs(d_vec[xIndex]) > 1e-5 ){
			d_value[0] = d_vec[xIndex];
                }
        }
}

__global__ void createBandedStructure(float *d_vec, int *d_bin, int k_bin, int * band_size, float val, int n){
        unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
        if( (xIndex < n) & (d_bin[xIndex] <= k_bin) ){
                if(atomicSub(band_size, 1) > 0){
                        d_vec[xIndex] = val;
                }
        }
}


/*
**********************************
** MAIN FUNCTION (SMV): createVector_smv  **
**********************************
*/

void  createVector_smv(int * k_pointer, const int m, const int n, int vecDistribution, float band_percentage, float *d_vec_input, float *values_rand, float *support_rand)
{
/*
  hipEvent_t startHost, stopHost;
  float hostTime;
  hipEventCreate(&startHost);
  hipEventCreate(&stopHost);
  hipEventRecord(startHost,0);
*/

  int threads_perblock = min(n, 512);
  dim3 threadsPerBlock(threads_perblock);
  int num_blocks = (int)ceil((float)n/(float)threads_perblock);
  dim3 numBlocks(num_blocks);


  // determine the random support with fast binning

  
  float *bin_vec, *d_bin_counter;

  hipMalloc((void**)&bin_vec, n * sizeof(int));
  SAFEcudaMalloc("bin_vector in createVector");

  hipMalloc((void**)&d_bin_counter, n * sizeof(int)); 
  SAFEcudaMalloc("d_bin_counter in createVector");

  int * h_bin_counter = (int*)malloc( sizeof(int) * n );
  SAFEmalloc_int(h_bin_counter, "h_bin_counter in createVector");

  zero_vector_int<<< numBlocks, threadsPerBlock >>>((int*)bin_vec, n);
  SAFEcuda("zero_vector_int in createVector (d_bin)");
  //hipDeviceSynchronize();

  zero_vector_int<<< numBlocks, threadsPerBlock >>>((int*)d_bin_counter, n);
  SAFEcuda("zero_vector_int in createVector (d_bin_counter)");
  //hipDeviceSynchronize();

  zero_vector_float<<< numBlocks, threadsPerBlock >>>((float*)d_vec_input, n);
  SAFEcuda("zero_vector_float in createVector");
  //hipDeviceSynchronize();

  int ind_abs_max = hipblasIsamax(n, support_rand, 1) - 1;
  SAFEcublas("hipblasIsamax in createVector");

  float max_value;
  hipMemcpy(&max_value, support_rand + ind_abs_max, sizeof(float), hipMemcpyDeviceToHost);
  SAFEcuda("hipMemcpy max_value in createVector");

  max_value = abs(max_value);
  float slope = ((n-1)/max_value);

  LinearBinning <<< numBlocks, threadsPerBlock >>>((float*)support_rand, (int*)bin_vec, (int*)d_bin_counter, n, n, n, slope, max_value);
  SAFEcuda("LinearBinning in createVector");


  hipMemcpy(h_bin_counter, d_bin_counter, n * sizeof(int), hipMemcpyDeviceToHost);
  SAFEcuda("hipMemcpy h_bin_counter in createVector");

  int kk=*k_pointer;

  int k_bin = 0;
  int sum=0;

  while ( (sum<kk) & (k_bin<n) ) {
	sum = sum + h_bin_counter[k_bin];
	k_bin++;
	}
  k_bin = k_bin-1;

 // printf("Inside createVector, kbin is: %d \n", k_bin);

  free(h_bin_counter);
  hipFree(d_bin_counter);

  threshold_one<<< numBlocks, threadsPerBlock >>>((float*)values_rand, (float*)d_vec_input, (int*)bin_vec, k_bin, n);
  SAFEcuda("threshold_one in createVector");

  *k_pointer = sum;

  //hipDeviceSynchronize();

  if (0 < band_percentage & band_percentage <= 1){
        //printf("band_percentage = %g\n", band_percentage);
        int k = *k_pointer;
	int band_size = min((int) (( (float) k )*band_percentage), k);
        //printf("band_size = %d\n", band_size);
	int * d_band_size;
        hipMalloc((void**)&d_band_size, 1 * sizeof(int));
	hipMemcpy(d_band_size, &band_size, sizeof(int), hipMemcpyHostToDevice);

	float * d_value;
	float zero = 0.0;
        hipMalloc((void**)&d_value, 1 * sizeof(float));
	hipMemcpy(d_value, &zero, sizeof(float), hipMemcpyHostToDevice);
	lookForValue<<<numBlocks, threadsPerBlock>>>((float*)d_vec_input, (float *)d_value, n);

	float h_value = 0.0;
	hipMemcpy(&h_value, d_value, sizeof(float), hipMemcpyDeviceToHost);
	//printf("h_value = %g\n", h_value);

	//createBandedStructure(float *d_vec, int *d_bin, int k_bin, int * band_size, float val, int n){
	//printf("About to createBandedStructure\n");
	createBandedStructure<<< numBlocks, threadsPerBlock >>>((float*)d_vec_input, (int*)bin_vec, k_bin, d_band_size, h_value, n);

	// Check banded Structure
	/*
	float value = 0;
	int bin = 0;
	for (int i = 0; i < n; i++){
		hipMemcpy(&bin, bin_vec+i, sizeof(int), hipMemcpyDeviceToHost);
		if (bin <= k_bin){
			hipMemcpy(&value, d_vec_input+i, sizeof(int), hipMemcpyDeviceToHost);
			printf("%g -> ", value);
		}
	}
	printf("\n", value);
	*/

  }
 
/*
// ******  checking that there is something in the vector *******

  float * h_vec_input = (float*)malloc( sizeof(float) * n );
  hipMemcpy(h_vec_input, d_vec_input, n * sizeof(float), hipMemcpyDeviceToHost);
  printf(" the first 25 entries of the new input vector: \n");
  for (j = 0; j<25; j++) printf(" %f ", h_vec_input[j]);
  printf("\n");

  free(h_vec_input);
*/

  hipFree(bin_vec);

/*
  hipDeviceSynchronize();
  hipEventRecord(stopHost,0);
  hipEventSynchronize(stopHost);
  hipEventElapsedTime(&hostTime, startHost, stopHost);
  hipEventDestroy(startHost);
  hipEventDestroy(stopHost);

  printf("The function createVector takes %f ms.\n", hostTime);
*/

  return;
}



/*
**********************************
** MAIN FUNCTION: createVector  **
**********************************
*/

void  createVector(int * k_pointer, const int m, const int n, int vecDistribution, float *d_vec_input, float *values_rand, float *support_rand)
{
/*
  hipEvent_t startHost, stopHost;
  float hostTime;
  hipEventCreate(&startHost);
  hipEventCreate(&stopHost);
  hipEventRecord(startHost,0);
*/

  int threads_perblock = min(n, 512);
  dim3 threadsPerBlock(threads_perblock);
  int num_blocks = (int)ceil((float)n/(float)threads_perblock);
  dim3 numBlocks(num_blocks);


  // determine the random support with fast binning

  
  float *bin_vec, *d_bin_counter;

  hipMalloc((void**)&bin_vec, n * sizeof(int));
  SAFEcudaMalloc("bin_vector in createVector");

  hipMalloc((void**)&d_bin_counter, n * sizeof(int)); 
  SAFEcudaMalloc("d_bin_counter in createVector");

  int * h_bin_counter = (int*)malloc( sizeof(int) * n );
  SAFEmalloc_int(h_bin_counter, "h_bin_counter in createVector");

  zero_vector_int<<< numBlocks, threadsPerBlock >>>((int*)bin_vec, n);
  SAFEcuda("zero_vector_int in createVector (d_bin)");
  //hipDeviceSynchronize();

  zero_vector_int<<< numBlocks, threadsPerBlock >>>((int*)d_bin_counter, n);
  SAFEcuda("zero_vector_int in createVector (d_bin_counter)");
  //hipDeviceSynchronize();

  zero_vector_float<<< numBlocks, threadsPerBlock >>>((float*)d_vec_input, n);
  SAFEcuda("zero_vector_float in createVector");
  //hipDeviceSynchronize();

  int ind_abs_max = hipblasIsamax(n, support_rand, 1) - 1;
  SAFEcublas("hipblasIsamax in createVector");

  float max_value;
  hipMemcpy(&max_value, support_rand + ind_abs_max, sizeof(float), hipMemcpyDeviceToHost);
  SAFEcuda("hipMemcpy max_value in createVector");

  max_value = abs(max_value);
  float slope = ((n-1)/max_value);

  LinearBinning <<< numBlocks, threadsPerBlock >>>((float*)support_rand, (int*)bin_vec, (int*)d_bin_counter, n, n, n, slope, max_value);
  SAFEcuda("LinearBinning in createVector");


  hipMemcpy(h_bin_counter, d_bin_counter, n * sizeof(int), hipMemcpyDeviceToHost);
  SAFEcuda("hipMemcpy h_bin_counter in createVector");

  int kk=*k_pointer;

  int k_bin = 0;
  int sum=0;

  while ( (sum<kk) & (k_bin<n) ) {
	sum = sum + h_bin_counter[k_bin];
	k_bin++;
	}
  k_bin = k_bin-1;

 // printf("Inside createVector, kbin is: %d \n", k_bin);

  free(h_bin_counter);
  hipFree(d_bin_counter);

  threshold_one<<< numBlocks, threadsPerBlock >>>((float*)values_rand, (float*)d_vec_input, (int*)bin_vec, k_bin, n);
  SAFEcuda("threshold_one in createVector");

  *k_pointer = sum;

  //hipDeviceSynchronize();

/*
// ******  checking that there is something in the vector *******

  float * h_vec_input = (float*)malloc( sizeof(float) * n );
  hipMemcpy(h_vec_input, d_vec_input, n * sizeof(float), hipMemcpyDeviceToHost);
  printf(" the first 25 entries of the new input vector: \n");
  for (j = 0; j<25; j++) printf(" %f ", h_vec_input[j]);
  printf("\n");

  free(h_vec_input);
*/

  hipFree(bin_vec);

/*
  hipDeviceSynchronize();
  hipEventRecord(stopHost,0);
  hipEventSynchronize(stopHost);
  hipEventElapsedTime(&hostTime, startHost, stopHost);
  hipEventDestroy(startHost);
  hipEventDestroy(stopHost);

  printf("The function createVector takes %f ms.\n", hostTime);
*/

  return;
}






