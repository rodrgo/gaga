#include "hip/hip_runtime.h"
/* Copyright 2010-2013 Jeffrey D. Blanchard and Jared Tanner
 *   
 * GPU Accelerated Greedy Algorithms for Compressed Sensing
 *
 * Licensed under the GAGA License available at gaga4cs.org and included as GAGA_license.txt.
 *
 * In  order to use the GAGA library, or any of its constituent parts, a user must
 * agree to abide by a set of * conditions of use. The library is available at no cost 
 * for ``Internal'' use. ``Internal'' use of the library * is defined to be use of the 
 * library by a person or institution for academic, educational, or research purposes 
 * under the conditions in the included GAGA_license.txt. Any use of the library implies 
 * that these conditions have been understood, and that the user agrees to abide by all 
 * the listed conditions.
 *     
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * Any redistribution or derivatives of this software must contain this header in all files
 * and include a copy of GAGA_license.txt.
 */



/*
**************************************
** MAIN FUNCTION: createProblem_gen **
**************************************
*/

void createProblem_gen(int *k_pointer, const int m, const int n, const int vecDistribution, float *d_vec_input, float *d_y, float *d_A, const int ensemble, unsigned int *p_seed)
{
// k_pointer is used rather than k as we may allow k to vary somewhat.
// m and n are the size of the matrix
// vecDistribution: 0 = uniform, 1=random{-1,1}, 2 = Gaussian
// d_vec_input is the vector the matrix multiplies "x"
// d_A is the matrix
// ensemble (matrix): 1=nonzeros all 1, 2=nonzeros random{-1,1}.
// p_seed allows us to store the seed and thus recreate the problem.

/*
  hipEvent_t startHost, stopHost;
  float hostTime;
  hipEventCreate(&startHost);
  hipEventCreate(&stopHost);
  hipEventRecord(startHost,0);
*/

  // To create this problem we need
  //	n random numbers to determine the random values of the input vector
  //	n random numbers to determine the support of the input vector
  //	n*m random numbers for the values in d_A


  // ensemble=1 is gaussian
  // ensemble=2 is random {-1,1}

  int L = 2*n; // we don't compute all random numbers at once because most could 
               // are the entries in d_A, which could be gaussian or \pm 1.


// ********* Create L random numbers needed to generate the problem ******


  // Allocate device variable for random numbers
  float * random;
  hipMalloc((void**)&random, L * sizeof(float));
  SAFEcudaMalloc("random in createProblem_gen");

  hiprandStatus_t curandCheck;

  hiprandGenerator_t gen;
  curandCheck = hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  SAFEcurand(curandCheck, "hiprandCreateGenerator in createProblem_gen");
  // note, HIPRAND_RNG_PSEUDO_DEFAULT selects the random number generator type
  curandCheck = hiprandSetPseudoRandomGeneratorSeed(gen,*p_seed);
  SAFEcurand(curandCheck, "curandSet...Seed in createProblem_gen");
  curandCheck = hiprandGenerateUniform(gen,random,L);
  SAFEcurand(curandCheck, "hiprandGenerateUniform in createProblem_gen");
  SAFEcuda("createRandomNumbers uniform in createProblem_gen"); 


  // if vecDistribution == 1 then need uniform sign pattern for the
  // nonzeros in the vector measured.
  if(vecDistribution == 1){

        hipDeviceProp_t dp;
        hipGetDeviceProperties(&dp,0);
        unsigned int max_threads_per_block = dp.maxThreadsPerBlock;

        int threads_perblock = min(n, max_threads_per_block);
        dim3 threadsPerBlock(threads_perblock);
        int num_blocks = (int)ceil((float)n/(float)threads_perblock);
        dim3 numBlocks(num_blocks);

        sign_pattern<<< numBlocks, threadsPerBlock >>>(random,n);
        SAFEcuda("sign_pattern in createProblem_gen");
  }

  // if vecDistribution == 2 then need gaussian random numbers for the
  // nonzeros in the vector measured.
  if(vecDistribution == 2){       

        curandCheck = hiprandGenerateNormal(gen,random,n,0,1);
  	SAFEcurand(curandCheck, "hiprandGenerateNormal in createProblem_gen");
  }


  curandCheck = hiprandDestroyGenerator(gen);
  SAFEcurand(curandCheck, "hiprandDestroyGenerator in createProblem_gen");



/*
  The random numbers are used to create various things:
  values_rand = random;
  support_rand = random + n;
*/
  createVector(k_pointer, m, n, vecDistribution, d_vec_input, random, random+n);
  SAFEcuda("createVector in createProblem_gen"); 

  createMeasurements_gen(m, n, d_vec_input, d_y, d_A, ensemble, p_seed);
  SAFEcuda("createMeasurements_gen in createProblem_gen"); 

  hipFree(random);
  SAFEcuda("hipFree random in createProblem_gen");



/*
  hipDeviceSynchronize();
  hipEventRecord(stopHost,0);
  hipEventSynchronize(stopHost);
  hipEventElapsedTime(&hostTime, startHost, stopHost);
  hipEventDestroy(startHost);
  hipEventDestroy(stopHost);

  printf("The function createProblem takes %f ms.\n", hostTime);
*/

}



void createProblem_gen_noise(int *k_pointer, const int m, const int n, const int vecDistribution, float *d_vec_input, float *d_y, float *d_A, const int ensemble, unsigned int *p_seed, float noise_level)
{
// k_pointer is used rather than k as we may allow k to vary somewhat.
// m and n are the size of the matrix
// vecDistribution: 0 = uniform, 1=random{-1,1}, 2 = Gaussian
// d_vec_input is the vector the matrix multiplies "x"
// d_A is the matrix
// ensemble (matrix): 1=nonzeros all 1, 2=nonzeros random{-1,1}.
// p_seed allows us to store the seed and thus recreate the problem.

/*
  hipEvent_t startHost, stopHost;
  float hostTime;
  hipEventCreate(&startHost);
  hipEventCreate(&stopHost);
  hipEventRecord(startHost,0);
*/

  // To create this problem we need
  //	n random numbers to determine the random values of the input vector
  //	n random numbers to determine the support of the input vector
  //	n*m entries for d_A
  //    and m entries for gaussian noise.
  //    Since hiprandGenerateNormal requires an even length, we add m mod 2.


  // ensemble=1 is gaussian
  // ensemble=2 is random {-1,1}

  int L = 2*n; // only compute the random numbers needed to make the vector and noise
  L=L+(m + m%2); // extra m entries for noise


// ********* Create L random numbers needed to generate the problem ******


  // Allocate device variable for random numbers
  float * random;
  hipMalloc((void**)&random, L * sizeof(float));
  SAFEcudaMalloc("random in createProblem_gen_noise");

  hiprandStatus_t curandCheck;

  hiprandGenerator_t gen;
  curandCheck = hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  SAFEcurand(curandCheck, "hiprandCreateGenerator in createProblem_gen_noise");
  // note, HIPRAND_RNG_PSEUDO_DEFAULT selects the random number generator type
  curandCheck = hiprandSetPseudoRandomGeneratorSeed(gen,*p_seed);
  SAFEcurand(curandCheck, "curandSet...Seed in createProblem_gen_noise");
  curandCheck = hiprandGenerateUniform(gen,random,L-m);
  SAFEcurand(curandCheck, "curandGenerateUnifrom in createProblem_gen_noise");
  curandCheck = hiprandGenerateNormal(gen,random+L-(m+m%2),m+m%2,0,1);
  SAFEcurand(curandCheck, "hiprandGenerateNormal (1st) in createProblem_gen_noise");
  SAFEcuda("createRandomNumbers in createProblem_gen_noise"); 


  // if vecDistribution == 1 then need uniform sign pattern for the
  // nonzeros in the vector measured.
  if(vecDistribution == 1){

        hipDeviceProp_t dp;
        hipGetDeviceProperties(&dp,0);
        unsigned int max_threads_per_block = dp.maxThreadsPerBlock;

        int threads_perblock = min(n, max_threads_per_block);
        dim3 threadsPerBlock(threads_perblock);
        int num_blocks = (int)ceil((float)n/(float)threads_perblock);
        dim3 numBlocks(num_blocks);

        sign_pattern<<< numBlocks, threadsPerBlock >>>(random,n);
        SAFEcuda("sign_pattern in createProblem_gen_noise");
  }

  // if vecDistribution == 2 then need gaussian random numbers for the
  // nonzeros in the vector measured.
  if(vecDistribution == 2){       

        curandCheck = hiprandGenerateNormal(gen,random,n,0,1);
  	SAFEcurand(curandCheck, "hiprandGenerateNormal (2nd) in createProblem_gen_noise");
  }

  curandCheck = hiprandDestroyGenerator(gen);
  SAFEcurand(curandCheck, "hiprandDestroyGenerator in createProblem_gen_noise");


/*
  The random numbers are used to create various things:
  values_rand = random;
  support_rand = random + n;
  noise = random + L - (m+m%2);
*/
  createVector(k_pointer, m, n, vecDistribution, d_vec_input, random, random+n);
  SAFEcuda("createVector in createProblem_gen_noise"); 

  createMeasurements_gen(m, n, d_vec_input, d_y, d_A, ensemble, p_seed);
  SAFEcuda("createMeasurements_gen in createProblem_gen_noise"); 

//  Here we want to add a noise vector that is scaled so that it's norm is noise_level*norm2(y).
  float noise_norm, y_norm, noise_scale;
  noise_norm = hipblasSnrm2(m, random+L-(m+m%2), 1);
  SAFEcublas("hipblasSnrm2 computing noise_norm in createProblem_gen_noise");

  y_norm = hipblasSnrm2(m, d_y, 1);
  SAFEcublas("hipblasSnrm2 computing y_norm in createProblem_gen_noise");

  noise_scale = noise_level*(y_norm/noise_norm);

  hipblasSaxpy(m, noise_scale, random+L-(m+m%2), 1, d_y, 1);
  SAFEcublas("hipblasSaxpy adding noise to measurements in createProblem_gen_noise");

  hipFree(random);
  SAFEcuda("hipFree random in createProblem_gen_noise");


/*
  hipDeviceSynchronize();
  hipEventRecord(stopHost,0);
  hipEventSynchronize(stopHost);
  hipEventElapsedTime(&hostTime, startHost, stopHost);
  hipEventDestroy(startHost);
  hipEventDestroy(stopHost);

  printf("The function createProblem takes %f ms.\n", hostTime);
*/

}

