#include "hip/hip_runtime.h"

/*
**************************************************
**		robust-l0			**
**  Noise robust version to parallel_l0		**
**************************************************
*/

inline void robust_l0(float *d_vec, float *d_y, float *d_res, int *d_rows, int *d_cols, float *d_vals, int *d_bin, int *d_bin_counters, int *h_bin_counters, const int num_bins, int *p_sum, float tol, const int maxiter, const int k, const int m, const int n, const int d, const int alpha, const int nz, float noise_level, float *resRecord, float *timeRecord, int *p_iter, dim3 numBlocks, dim3 threadsPerBlock, dim3 numBlocksnp, dim3 threadsPerBlocknp, dim3 numBlocksm, dim3 threadsPerBlockm, dim3 numBlocks_bin, dim3 threadsPerBlock_bin){

	int iter = *p_iter;
	int offset = 0;
	timeRecord[0] = 0.0;

	// Options/Inputs in robust_l0
	int enforce_l1_decrease = 0;
	int do_hard_thresholding = 1; // Might be redundant after changes
	int seed = 1111;

	float h_sigma_noise = noise_level;
	float h_sigma_s = 1.0;

	// Thresholding variables
	int k_bin = 0;
	float alpha_ht = 0.25f;
	int MaxBin = (int)(num_bins * (1 - alpha_ht));
	float minVal = 0.0f;
	float maxChange = 1.0f;
	float max_value = MaxMagnitude(d_vec, n);
	float slope = ((num_bins - 1)/(max_value));
	
	float * d_Ax;
	float * d_updates;
	
	hipMalloc((void**)&d_Ax, m * sizeof(float));
	hipMalloc((void**)&d_updates, n * sizeof(float));

	computeResidual(d_res, d_y, d_Ax, d_vec, d_rows, d_cols, d_vals, nz, m, n,
		 numBlocksm, threadsPerBlockm, numBlocksnp, threadsPerBlocknp);
	
	float norm_res;
	float norm_res_start;
	norm_res = hipblasSnrm2(m, d_res, 1);
	norm_res_start = norm_res;
	resRecord[0] = norm_res;

	int isCycling = 0;

	float *resCycling;
	int resCyclingLength = 2*d;
	int resRepeatLength = d - 1;
	resCycling = (float*) malloc(sizeof(float)*resCyclingLength);
	for (int i = 0; i < resCyclingLength; i++)
		resCycling[i] = 0.0;

	float *residNorm_prev;
	float *residNorm_evolution;
	float residNorm_diff = 1.0f;
	int residNorm_length = 2*d;
	residNorm_prev = (float*) malloc(sizeof(float)*residNorm_length);
	residNorm_evolution = (float*) malloc(sizeof(float)*residNorm_length);
	for (int i = 0; i < residNorm_length; i++){
		residNorm_prev[i] = 0.0;
		residNorm_evolution[i] = 1.0f;
	}

        // hiprand: uniform random variables
	float * d_unif;
	int d_unif_size = (2*d + 1)*n;
	hipMalloc((void**)&d_unif, d_unif_size*sizeof(float));

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	//hiprandGenerateUniform(gen, d_unif, d_unif_size);

        // CUDA constants for p.d.f. construction
	float rho = ((float)k)/((float) m);

	float h_prob_zero_factor = expf(d*rho) - 1;
	float h_prob_equal_factor = expf(2*d*rho) - 1;
	float h_sigma_signal_zero = sqrtf(h_sigma_noise*h_sigma_noise + h_sigma_s*h_sigma_s*d*rho/(1- expf(-d*rho)));
	float h_sigma_signal_equal = sqrtf(2*h_sigma_noise*h_sigma_noise + 2*h_sigma_s*h_sigma_s*d*rho/(1 - expf(-2*d*rho)));

	hipMemcpyToSymbol(HIP_SYMBOL(sigma_noise), &h_sigma_noise, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(prob_zero_factor), &h_prob_zero_factor, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(prob_equal_factor), &h_prob_equal_factor, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sigma_signal_zero), &h_sigma_signal_zero, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sigma_signal_equal), &h_sigma_signal_equal, sizeof(float));

	int *d_bernoulli_pz_u;
	int *d_scores;

	hipMalloc((void**)&d_bernoulli_pz_u, n*sizeof(int));
	hipMalloc((void**)&d_scores, n*sizeof(int));

	float *d_average_updates;
	hipMalloc((void**)&d_average_updates, n*sizeof(float));

	float norm_res_mean = (h_sigma_noise*h_sigma_noise)*((float) m);
	float norm_res_sd = (h_sigma_noise*h_sigma_noise)*sqrtf(2 * ((float) m)); 

	while( (iter < maxiter) & (norm_res*norm_res - norm_res_mean > tol*norm_res_sd) & (norm_res < (100*norm_res_start)) & (residNorm_diff > 0.000001) & (isCycling == 0)){

		// time variables
		hipEvent_t start, stop;
		float time;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		// BEGIN STEP
		offset = iter % d;
		
		// generate uniform random variables
		// d_unif_size = (2*d + 1)*n
		hiprandGenerateUniform(gen, d_unif, d_unif_size);
		hipDeviceSynchronize();
		
		// compute l0-scores in parallel
		// d_unif has "n" chunks of size "2*d + 1"
		zero_vector_float<<<numBlocks, threadsPerBlock>>>(d_average_updates, n);

		// Takes "1" uniform random variable.
		cuda_sample_prob_zero_u<<<numBlocks, threadsPerBlock>>>(d_bernoulli_pz_u, d_unif, d, n, d_res, d_rows, offset);
		hipDeviceSynchronize();

		// Compute scores
		cuda_compute_scores<<<numBlocks, threadsPerBlock>>>(d_scores, d_bernoulli_pz_u, d_average_updates, d_unif, d, n, d_res, d_rows, offset);
		hipDeviceSynchronize();

		// Compute update signal robust l0
		cuda_update_signal_robust_l0<<<numBlocks, threadsPerBlock>>>(d_res, d_bernoulli_pz_u, d_vec, d_updates, d_average_updates, d_rows, n, d, alpha, d_scores, enforce_l1_decrease); 
		hipDeviceSynchronize();

		// compute residual
		computeResidual(d_res, d_y, d_Ax, d_vec, d_rows, d_cols, d_vals, nz, m, n,
			 numBlocksm, threadsPerBlockm, numBlocksnp, threadsPerBlocknp);
		hipDeviceSynchronize();

		//cuda_update_residual<<<numBlocks, threadsPerBlock>>>(d_res, d_rows, d_average_updates, n, d);

		//cuda_update_residual_np<<<numBlocksnp, threadsPerBlocknp>>>(d_res, d_rows, d_updates, n, d);

		if (do_hard_thresholding == 1){
			// thresholding
			H_k(d_vec, k, n, d_bin, d_bin_counters, h_bin_counters, &maxChange, &max_value,
				&slope, &minVal, &alpha_ht, &MaxBin, &k_bin, p_sum, num_bins,
				numBlocks, threadsPerBlock, numBlocks_bin, threadsPerBlock_bin);

			// recompute residual
			computeResidual(d_res, d_y, d_Ax, d_vec, d_rows, d_cols, d_vals, nz, m, n, 
				numBlocksm, threadsPerBlockm, numBlocksnp, threadsPerBlocknp);
		}

		norm_res = hipblasSnrm2(m, d_res, 1);

		// END STEP

		// check for no change in residual 
		for (int i = 0; i < residNorm_length - 1; i++){
			residNorm_prev[i] = residNorm_prev[i + 1];
		}
		residNorm_prev[residNorm_length - 1] = norm_res;
		for (int i = 0; i < residNorm_length - 1; i++){
			residNorm_evolution[i] = residNorm_evolution[i + 1];
		}
		residNorm_evolution[residNorm_length - 1] = residNorm_prev[residNorm_length - 2] - residNorm_prev[residNorm_length - 1];
		residNorm_diff = max_list(residNorm_evolution, residNorm_length);


		// Check for cycling
		// Check works, but not in all cases. Need more precise check.
		for (int i = 0; i < resCyclingLength - 1; i++)
			resCycling[i] = resCycling[i + 1]; 
		resCycling[resCyclingLength - 1] = norm_res;
		if (iter > resCyclingLength){
			isCycling = 1;
			for (int i = 3; i < resCyclingLength; i = i + 2)
				isCycling = isCycling*(resCycling[i] == resCycling[i - 2]);
		}
		if (iter > resRepeatLength){
			isCycling = 1;
			for (int i = resCyclingLength - 1; i >= resCyclingLength - resRepeatLength; i = i - 1)
				isCycling = isCycling*(abs(resCycling[i] - resCycling[i-1]) <= 1e-10);

		}

		// end timing
		hipDeviceSynchronize();
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);
	
		iter = iter + 1;
		resRecord[iter] = norm_res;
		timeRecord[iter] = timeRecord[iter - 1] + time;

		//printf("iter = %d, norm_res = %5.6f, isCycling = %d\n", iter, norm_res, isCycling);

	}

	float norm_xhat = 0.0;
	float big_float = 10000.0;
	
	norm_xhat = hipblasSnrm2(n, d_vec, 1);

	// If norm_xhat == 0.0, then no updates were performed.
	// We increase the norm so that GAGA doesn't think it is a success (when n is large)
	if (norm_xhat == 0.0){
		hipMemcpy(d_vec, &big_float, sizeof(float), hipMemcpyHostToDevice);
	}

	*p_iter = iter;

	free(resCycling);
	free(residNorm_prev);
	free(residNorm_evolution);

	// clean GPU
	hipFree(d_Ax);
	hipFree(d_updates);
        //hipFree(randArray);
        //hipFree(d_state);

	hipFree(d_average_updates);
	hipFree(d_unif);
	hipFree(d_bernoulli_pz_u);
	hipFree(d_scores);

}

