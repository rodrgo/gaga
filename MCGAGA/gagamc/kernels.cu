
#include <hip/hip_runtime.h>
/* Copyright 2010-2013 Jeffrey D. Blanchard and Jared Tanner
 *   
 * GPU Accelerated Greedy Algorithms for Compressed Sensing
 *
 * Licensed under the GAGA License available at gaga4cs.org and included as GAGA_license.txt.
 *
 * In  order to use the GAGA library, or any of its constituent parts, a user must
 * agree to abide by a set of * conditions of use. The library is available at no cost 
 * for ``Internal'' use. ``Internal'' use of the library * is defined to be use of the 
 * library by a person or institution for academic, educational, or research purposes 
 * under the conditions in the included GAGA_license.txt. Any use of the library implies 
 * that these conditions have been understood, and that the user agrees to abide by all 
 * the listed conditions.
 *     
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * Any redistribution or derivatives of this software must contain this header in all files
 * and include a copy of GAGA_license.txt.
 */






__global__ void zero_vector_float(float *vec, const int n)
{
  unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
  if ( xIndex < n )
    vec[xIndex]=0.0f;
}



__global__ void SVt_mult(float *V, float *S, const int n, const int nr)
{
/* This kernel performs the multiplication S*V^T where S is a diagonal matrix 
stored as a vector of r scalars and V is an n x r matrix stored in column major form. */
  unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
  if ( xIndex < nr )
    V[xIndex]*=S[(int)(xIndex/n)];
}
 
