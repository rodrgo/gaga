#include "hip/hip_runtime.h"
/* Copyright 2014 Jeffrey D. Blanchard and Jared Tanner
 *   
 * GPU Accelerated Greedy Algorithms for Matrix Completion
 *
 * Licensed under the GAGAMC License available at gaga4cs.org and included as GAGAMC_license.txt.
 *
 * In  order to use the GAGAMC library, or any of its constituent parts, a user must
 * agree to abide by a set of conditions of use. The library is available at no cost 
 * for ``Internal'' use. ``Internal'' use of the library is defined to be use of the 
 * library by a person or institution for academic, educational, or research purposes 
 * under the conditions in the included GAGA_license.txt. Any use of the library implies 
 * that these conditions have been understood, and that the user agrees to abide by all 
 * the listed conditions.
 *     
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * Any redistribution or derivatives of this software must contain this header in all files
 * and include a copy of GAGAMC_license.txt.
 */



#include "greedyheader.cu"


/*
*********** VERBOSE or SAFE ***************
**     IF you want verbose or safe,      **
**     change it in greedyheader.cu      **
*******************************************
*/




// Host function
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{

#ifdef VERBOSE
 cout <<"This is gagamc_entry_suic."<<endl;
#endif
//hipDeviceReset();
//SAFEcuda("DeviceReset");

  if ( (nlhs!=3)  && (nlhs!=7)  )
    printf("[gaga_gen] Error: There are two possible usages for this funtion.\n Five or six (with options) input arguments with six output arguments. \n [norms times iterations convRate MatOut MatInput A] = gagamc_entry(algstring,k,m,n,options).\n Five or six (with options) input arguments with three output arguments.\n [outputMatrix iterations convRate] = gagamc_entry(algstring,y,A,k,options).\n");
  else {


// reading in the string to determine the algorithm
    int strlen = mxGetN(prhs[0])+1;
    char algstr[strlen+100];
    int algerr = mxGetString(prhs[0], algstr, strlen);

// creating the switch variable defining the algorithm
    int alg;

// check that algstr indicates one of the valid algorithms:
// NIHT, HTP, IHT, ThresholdSD, ThresholdCG, CSMPSP, CGIHT
    int valid_alg = 0;
    if ( (strcmp(algstr, "NIHT")==0) || (strcmp(algstr, "SVT")==0) || (strcmp(algstr, "CGIHT")==0) || (strcmp(algstr, "LMaFit")==0)) valid_alg = 1;


// possible inputs
    int m, n, r, p;    // problem parameters p measurements of a rank r matrix of size m x n.
    float *h_Y, *h_y;
    int *h_A_in;          //  problem information with entry measurements in matrix and vector format, plust the entry sensing operator A 

// possible outputs
    int *h_A_out, *total_iter;
    float *h_norms, *h_times, *convergence_rate, *h_Mat_out, *h_Mat_input;

// control parameter from inputs
    int mn, mr, nr;

// make variables to store properties of the GPU (device)
    unsigned int max_threads_per_block;
    hipDeviceProp_t dp;
    


    if (valid_alg == 0){
      printf("[gaga_gen] Error: The possible input strings for algorithms using gaga_gen are:\n NIHT \n CGIHT \n SVT\n LMaFit \n");
    }
    else {

      if (nlhs == 7){
        m = (int)mxGetScalar(prhs[1]);
   	n = (int)mxGetScalar(prhs[2]);
   	r = (int)mxGetScalar(prhs[3]);
   	p = (int)mxGetScalar(prhs[4]);
   
				mn = m * n;
        mr = m * r;
        nr = n * r;

   	plhs[0] = mxCreateNumericMatrix(3, 1, mxSINGLE_CLASS, mxREAL);
   	h_norms = (float*) mxGetData(plhs[0]);

 	plhs[1] = mxCreateNumericMatrix(3, 1, mxSINGLE_CLASS, mxREAL);
   	h_times = (float*) mxGetData(plhs[1]);

 	plhs[2] = mxCreateNumericMatrix(1,1, mxINT32_CLASS, mxREAL);
   	total_iter = (int*) mxGetData(plhs[2]);

	plhs[3] = mxCreateNumericMatrix(1,1, mxSINGLE_CLASS, mxREAL);
   	convergence_rate = (float*) mxGetData(plhs[3]);

  	plhs[4] = mxCreateNumericMatrix(m, n, mxSINGLE_CLASS, mxREAL);
   	h_Mat_out = (float*) mxGetData(plhs[4]);

  	plhs[5] = mxCreateNumericMatrix(m, n, mxSINGLE_CLASS, mxREAL);
   	h_Mat_input = (float*) mxGetData(plhs[5]);

  	plhs[6] = mxCreateNumericMatrix(p, 1, mxINT32_CLASS, mxREAL);
   	h_A_out = (int*) mxGetData(plhs[6]);
      } // ends if (nlhs == 7)
      else if (nlhs == 3){

        h_Y = (float*)mxGetData(prhs[1]);
        h_y = (float*)mxGetData(prhs[2]);
        h_A_in = (int*)mxGetData(prhs[3]);
        r = (int)mxGetScalar(prhs[4]);

        m = (int)mxGetM(prhs[1]);
        n = (int)mxGetN(prhs[1]);
        p = (int)mxGetM(prhs[2]);

        mn = m * n;
        mr = m * r;
        nr = n * r;

    
        plhs[0] = mxCreateNumericMatrix(m, n, mxSINGLE_CLASS, mxREAL);
        h_Mat_out = (float*) mxGetData(plhs[0]);  
    
        plhs[1] = mxCreateNumericMatrix(1,1, mxINT32_CLASS, mxREAL);
        total_iter = (int*) mxGetData(plhs[1]);
    
        plhs[2] = mxCreateNumericMatrix(1,1, mxSINGLE_CLASS, mxREAL);
        convergence_rate = (float*) mxGetData(plhs[2]);
      }  // ends else if (nlhs == 3)


// possible options include:
    // tol: specifying convergence rate stopping conditions 
    // 	    (default=0.001)
    // maxiter: maximum number of iterations 
    // 		(default 300 for HTP and CSMPSP, 5000 for others)
    // vecDistribution: distribution of the nonzeros in the sparse vector for the test problem instance 
    // 	   		(default 'binary' indicating random plus or minus 1)
    // matrixEnsemble: distribution of the nonzeros in the measurement matrix for the test problem
    // 		       (default 'gaussian' indicating normal N(0,1))
    // seed: seed for random number generator
    //       (default set by clock())
    // numBins: number of bins to use for order statistics
    // 		(default set to max(n/20,1000))
    // threadsPerBlockn: number of threads per block for kernels acting on n vectors
    // 			 (default set to min(n, max_threads_per_block))
    // threadsPerBlockm: number of threads per block for kernels acting on m vectors
    // 			 (default set to min(m, max_threads_per_block))
    // threadsPerBlockBin: number of threads per block for kernels acting on vectors of length numBins
    // 			   (default set to min(num_bins, max_threads_per_block))
    // convRateNum: number of the last iterations to use when calculating average convergence rate
    // 		    (default set to 16)
    // kFixed: flag to force the k used in the problem generate to be that specified
    // 	       (default set to 'off')
    // noise: level of noise as a fraction of the \|Ax\|_2
    // 	      (default set to 0)
    // gpuNumber: which gpu to run the code on
    // 		  (default set to 0)
    // timing: indicates that times per iteration should be recorded
    // 	       (default = 'off')
    // alpha: specifying fraction of k used in early support set identification steps
    // 	      (default set to 0.25)
    // supportFlag: method by which the support set is identified 
    // 		     (default set to 0 for dynamic binning)
    // restartFlag: flag to decide an algorithm should restart some aspect, such as CG in CGIHT
    // 		    (default = "off")
    // projFracTol: tolerence for CGIHTprojected (default == 3);

// initialise options at default values
    // some of these may not be used depending on the usage (such as vecDistribution)
//    int vecDistribution = 1;  // binary
    int matrixEnsemble = 1;  // gaussian
//    int kFixedFlag = 0; // k not fixed
    unsigned int seed = clock();
//    int num_bins = max(n/20,1000);
    int gpuNumber = 1;
    int convRateNum = 16;
    float tol = 10^(-4);
    float PSVDtol = 0.01;
//    float noise_level = 0.0;
//    int timingFlag = 0; // off
//    float alpha_start = 0.25;
 //   int supp_flag = 0;
    int maxiter=50;
    int PSVDmaxiter = 15; 
    int restartFlag = 0; 
    float projFracTol = 3.0;
    if ( (strcmp(algstr, "HTP")==0) || (strcmp(algstr, "CSMPSP")==0) ) maxiter=300;
    else maxiter=5000;
// unlike other options, the threads_perblock options must be set to default
    // only after the option gpuNumber is determined when checking the options list
    int threads_perblockmn = 0; // min(n, max_threads_per_block);
    int threads_perblockm = 0; // min(m, max_threads_per_block);
    int threads_perblockp = 0; // min(p, max_threads_per_block);
    int threads_perblocknr = 0; // min(nr, max_threads_per_block);
		int threads_perblockmr = 0; //added by Chenshuai Sui
          
// set the gpu device properties in  case gpuNumber wasn't an option
    
    hipSetDevice(gpuNumber);
    SAFEcuda("hipSetDevice");
    hipGetDeviceProperties(&dp,gpuNumber);
    SAFEcuda("hipGetDeviceProperties");
    
    max_threads_per_block = dp.maxThreadsPerBlock;
		//printf("maxthreadsperblock = %i \n",  max_threads_per_block);
// extract the options if the last input argument is a cell.
    if ( mxIsCell(prhs[nrhs-1]) ){
      // set values for those options that have been specified
      if ( mxGetN(prhs[nrhs-1])==2 ){  // checking that the options list has two columns
        int numOptions = mxGetM(prhs[nrhs-1]);

	int nsubs = 2;
	int index, buflen;
	mxArray  *cell_element_ptr;
        for (int i=0; i< numOptions; i++){
	  // get the index of the i^th row, 1st column of the options cell
          int subs[]={i, 0};
	  index = mxCalcSingleSubscript(prhs[nrhs-1], nsubs, subs);
	  cell_element_ptr = mxGetCell(prhs[nrhs-1], index);
	  buflen = (mxGetM(cell_element_ptr) * 
                    mxGetN(cell_element_ptr)) + 1;
	  char *buf = (char*) malloc(buflen);
      	  algerr = mxGetString(cell_element_ptr, buf, buflen);

	  // go through the list of possible options looking for match with buf
	  subs[1]=1; // move to second column
	  index = mxCalcSingleSubscript(prhs[nrhs-1], nsubs, subs);
    	  cell_element_ptr = mxGetCell(prhs[nrhs-1], index);

	  if (strcmp(buf, "tol")==0){
	    float *p_num = (float*) mxGetData(cell_element_ptr);
	    tol = *p_num; }
	  else if (strcmp(buf, "maxiter")==0){
	    int *p_num = (int*) mxGetData(cell_element_ptr);
	    maxiter = *p_num; }
	  else if (strcmp(buf, "PSVDtol")==0){
	    float *p_num = (float*) mxGetData(cell_element_ptr);
	    PSVDtol = *p_num; }
	  else if (strcmp(buf, "PSVDmaxiter")==0){
	    int *p_num = (int*) mxGetData(cell_element_ptr);
	    PSVDmaxiter = *p_num; }
          else if (strcmp(buf, "projFracTol")==0) { 
            float *p_num = (float*) mxGetData(cell_element_ptr);
            projFracTol = *p_num; }
	  else if (strcmp(buf, "gpuNumber")==0){
	    int *p_num = (int*) mxGetData(cell_element_ptr);
	    gpuNumber = *p_num; 
	    int devCount;
	    hipGetDeviceCount(&devCount);
            SAFEcuda("DeviceCount");
   	    if ((gpuNumber >= devCount) && (gpuNumber != 0)){
     	      cout << "This computer has " << devCount 
      	      	   << " gpus and gpuNumber was" << endl << "selected at " 
	   	   << gpuNumber << " which is larger than admissible." 
	   	   << endl << "gpuNumber has been reset to 0." << endl; 
     	      gpuNumber = 0; }
	    hipSetDevice(gpuNumber);
            SAFEcuda("SetDevice");
	    hipGetDeviceProperties(&dp,gpuNumber);
            SAFEcuda("GetDeviceProperties");
    	    max_threads_per_block = dp.maxThreadsPerBlock; }
	  else if (strcmp(buf, "threadsPerBlockmn")==0){
	    int *p_num = (int*) mxGetData(cell_element_ptr);
	    threads_perblockmn = *p_num; }
	  else if (strcmp(buf, "threadsPerBlockm")==0){
	    int *p_num = (int*) mxGetData(cell_element_ptr);
	    threads_perblockm = *p_num; }
	  else if (strcmp(buf, "threadsPerBlockp")==0){
	    int *p_num = (int*) mxGetData(cell_element_ptr);
	    threads_perblockp = *p_num; }
	  else if (strcmp(buf, "threadsPerBlocknr")==0){
	    int *p_num = (int*) mxGetData(cell_element_ptr);
	    threads_perblocknr = *p_num; }
	  else if (strcmp(buf, "convRateNum")==0){
	    int *p_num = (int*) mxGetData(cell_element_ptr);
	    convRateNum = *p_num; }
	  else if (strcmp(buf, "seed")==0){
	    unsigned int *p_num = (unsigned int*) mxGetData(cell_element_ptr);
	    seed = *p_num; }
/*
	  else if (strcmp(buf, "noise")==0){
	    float *p_num = (float*) mxGetData(cell_element_ptr);
	    noise_level = *p_num; }
	  else if (strcmp(buf, "alpha")==0){
	    float *p_num = (float*) mxGetData(cell_element_ptr);
	    alpha_start = *p_num; }
	  else if (strcmp(buf, "supportFlag")==0){
	    int *p_num = (int*) mxGetData(cell_element_ptr);
	    supp_flag = *p_num; }
	  else if (strcmp(buf, "vecDistribution")==0){
	    int buflen_tmp = (mxGetM(cell_element_ptr) * 
                              mxGetN(cell_element_ptr)) + 1;
	    char *buf_tmp = (char*) malloc(buflen_tmp);
       	    algerr = mxGetString(cell_element_ptr, buf_tmp, buflen_tmp);
	    if (strcmp(buf_tmp, "binary")==0) vecDistribution = 1;
	    else if (strcmp(buf_tmp, "uniform")==0) vecDistribution = 0;
	    else if (strcmp(buf_tmp, "gaussian")==0) vecDistribution = 2;}
	  else if (strcmp(buf, "kFixed")==0){
	    int buflen_tmp = (mxGetM(cell_element_ptr) * 
                              mxGetN(cell_element_ptr)) + 1;
	    char *buf_tmp = (char*) malloc(buflen_tmp);
       	    algerr = mxGetString(cell_element_ptr, buf_tmp, buflen_tmp);
	    if (strcmp(buf_tmp, "on")==0) kFixedFlag = 1;
	    else if (strcmp(buf_tmp, "off")==0) kFixedFlag = 0;}
	  else if (strcmp(buf, "timing")==0){
	    int buflen_tmp = (mxGetM(cell_element_ptr) * 
                              mxGetN(cell_element_ptr)) + 1;
	    char *buf_tmp = (char*) malloc(buflen_tmp);
       	    algerr = mxGetString(cell_element_ptr, buf_tmp, buflen_tmp);
	    if (strcmp(buf_tmp, "on")==0) timingFlag = 1;
	    else if (strcmp(buf_tmp, "off")==0) timingFlag = 0;}
*/
	  else if (strcmp(buf, "restartFlag")==0){
	    int buflen_tmp = (mxGetM(cell_element_ptr) * 
                              mxGetN(cell_element_ptr)) + 1;
	    char *buf_tmp = (char*) malloc(buflen_tmp);
       	    algerr = mxGetString(cell_element_ptr, buf_tmp, buflen_tmp);
	    if (strcmp(buf_tmp, "on")==0) restartFlag = 1;
	    else if (strcmp(buf_tmp, "off")==0) restartFlag = 0;}
	  else if (strcmp(buf, "matrixEnsemble")==0){
	    int buflen_tmp = (mxGetM(cell_element_ptr) * 
                              mxGetN(cell_element_ptr)) + 1;
	    char *buf_tmp = (char*) malloc(buflen_tmp);
       	    algerr = mxGetString(cell_element_ptr, buf_tmp, buflen_tmp);
	    if (strcmp(buf_tmp, "binary")==0) matrixEnsemble = 2;
	    else if (strcmp(buf_tmp, "gaussian")==0) matrixEnsemble = 1;
	    else cout << "Admissible matrixEnsemble for gen are: binary and gaussian.\n";}
	  else{
	    cout << "The following option is not recognised: " << buf << endl;
	  }
	}
      }
    }



// check if any of the threads_perblock variable were not set in the options
    if (threads_perblockmn == 0) threads_perblockmn = min(mn, max_threads_per_block);
    if (threads_perblockm == 0) threads_perblockm = min(m, max_threads_per_block);
    if (threads_perblockp == 0) threads_perblockp = min(p, max_threads_per_block);
    if (threads_perblocknr == 0) threads_perblocknr = min(nr, max_threads_per_block);
		if (threads_perblockmr == 0) threads_perblockmr = min(mr, max_threads_per_block);  //added by Chenshuai Sui
/*
// output alert if timing is specified for an algorthm other than NIHT and HTP
   if ( timingFlag == 1 ){
     if ( !((strcmp(algstr, "NIHT")==0) || (strcmp(algstr, "HTP")==0)) )
       cout << "The timing option is only available for NIHT and HTP, using the non-timing variant.\n";
   }
*/
// generate variables for cuda timings
    hipEvent_t startTest, stopTest;
    hipEvent_t *p_startTest, *p_stopTest;
    p_startTest = &startTest;
    p_stopTest = &stopTest;
    hipEventCreate(p_startTest);
    SAFEcuda("EventCreate p_startTest");
    hipEventCreate(p_stopTest);
    SAFEcuda("EventCreate p_stopTest");
    hipEventRecord(startTest,0);
    SAFEcuda("EventRecord startTest");

// establish a generator and seed for cuRand to use in problem creation and partial SVD
    hiprandGenerator_t gen;
    hiprandStatus_t curandCheck;
    curandCheck = hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    SAFEcurand(curandCheck, "hiprandCreateGenerator in gagamc_entry_suic");
    // note, HIPRAND_RNG_PSEUDO_DEFAULT selects the random number generator type
    curandCheck = hiprandSetPseudoRandomGeneratorSeed(gen,seed);
    SAFEcurand(curandCheck, "curandSet...Seed in gagamc_entry_suic"); 



// Allocate variables on the device
    float * d_Mat;
    float * Grad;
    float * Grad_proj;
    float * Grad_prev;
    float * Grad_prev_proj;
    float * d_Y;
    float * d_U;
    float * d_S;
    float * d_V;
    float * d_y;
    float * d_y_work;
    float * d_u;
    float * d_u_prev;
    float * d_v;
    int * d_A;
    float * d_U_prev;		//needed in PartialSVD_SPI
    float * d_MM;				//needed in PartialSVD_SPI
    float *d_R;					//needed in PartialSVD_SPI
		float *d_U_init;  // needed in SVT
		float *d_V_prev;	//needed in LMaFit
		float *d_RR;			//needed in LMaFit
		float *d_SS;				//needed in LMaFit


		float *h_S;				//needed in PartialSVD_SPI
		float *h_S_prev;	//needed in PartialSVD_SPI

		

// allocate memory on the device 


    hipMalloc((void**)&d_Mat, mn * sizeof(float));
    SAFEcudaMalloc("d_Mat");

    hipMalloc((void**)&Grad, mn * sizeof(float));
    SAFEcudaMalloc("Grad");

    hipMalloc((void**)&Grad_proj, mn * sizeof(float));
    SAFEcudaMalloc("Grad_proj");
  
    hipMalloc((void**)&d_Y, mn * sizeof(float));
    SAFEcudaMalloc("d_Y");
  
    hipMalloc((void**)&d_y, p * sizeof(float));
    SAFEcudaMalloc("d_y");

    hipMalloc((void**)&d_u, m * sizeof(float));
    SAFEcudaMalloc("d_u");

    hipMalloc((void**)&d_u_prev, m * sizeof(float));
    SAFEcudaMalloc("d_u_prev");

    hipMalloc((void**)&d_v, n * sizeof(float));
    SAFEcudaMalloc("d_v");
  
    hipMalloc((void**)&d_A, p * sizeof(int));
    SAFEcudaMalloc("d_A");

    hipMalloc((void**)&d_MM, (m*m) * sizeof(int));
    SAFEcudaMalloc("d_MM"); 

		


    if (strcmp(algstr, "CGIHT")==0) {

      hipMalloc((void**)&Grad_prev, mn * sizeof(float));
      SAFEcudaMalloc("Grad_prev");

      hipMalloc((void**)&Grad_prev_proj, mn * sizeof(float));
      SAFEcudaMalloc("Grad_prev_proj");

      hipMalloc((void**)&d_y_work, p * sizeof(float));
      SAFEcudaMalloc("d_y_work");

    }

		if (strcmp(algstr, "SVT")==0 ){
			//allocate memory on device
			hipMalloc((void**)&d_U_init, mn * sizeof(float));
      SAFEcudaMalloc("d_U_init");
			curandCheck = hiprandGenerateNormal(gen, d_U_init, m*n, 0,1);
			SAFEcurand(curandCheck, "hiprandGenerateNormal in SVT.cu");

			hipMalloc((void**)&d_U, mn * sizeof(float));
    	SAFEcudaMalloc("d_U");

			hipMalloc((void**)&d_U_prev, mn * sizeof(float));
    	SAFEcudaMalloc("d_U_prev");
  
    	hipMalloc((void**)&d_S, std::min(m,n) * sizeof(float));
    	SAFEcudaMalloc("d_S");
  
    	hipMalloc((void**)&d_V, mn * sizeof(float));
    	SAFEcudaMalloc("d_V");

			hipMalloc((void**)&d_R,  std::min(m,n)* sizeof(int));
    	SAFEcudaMalloc("d_R"); 

			//allocate memory on host
			h_S = (float*)malloc( sizeof(float) * std::min(m,n));
    	SAFEmalloc_float(h_S, "h_S");

			h_S_prev = (float*)malloc( sizeof(float) * std::min(m,n));
    	SAFEmalloc_float(h_S_prev, "h_S_prev");
		}
		else{

			//allocate memory on device
			hipMalloc((void**)&d_U, mr * sizeof(float));
    	SAFEcudaMalloc("d_U");
  
    	hipMalloc((void**)&d_S, r * sizeof(float));
    	SAFEcudaMalloc("d_S");
  
    	hipMalloc((void**)&d_V, nr * sizeof(float));
    	SAFEcudaMalloc("d_V");

			hipMalloc((void**)&d_U_prev, mr * sizeof(int));
    	SAFEcudaMalloc("d_U_prev");

			hipMalloc((void**)&d_R, r * sizeof(int));
    	SAFEcudaMalloc("d_R");

			//allocate memory on host

			h_S = (float*)malloc( sizeof(float) * r );
    	SAFEmalloc_float(h_S, "h_S");

    	h_S_prev = (float*)malloc( sizeof(float) * r );
    	SAFEmalloc_float(h_S_prev, "h_S_prev");

		}

		if (strcmp(algstr, "LMaFit")==0 ){	
			hipMalloc((void**)&d_V_prev, nr * sizeof(int));
    	SAFEcudaMalloc("d_V_prev"); 
			hipMalloc((void**)&d_RR, (r*r) * sizeof(int));
    	SAFEcudaMalloc("d_RR");
			hipMalloc((void**)&d_SS, mn * sizeof(float));
    	SAFEcudaMalloc("d_SS");
		}


			

   
// allocate memory on the host

    float * residNorm_prev = (float*)malloc( sizeof(float) * convRateNum );
    SAFEmalloc_float(residNorm_prev, "residNorm_prev");

		int r_estimate;
		int * p_r = &r_estimate;  // to store rank estimate



// allocate memory on the host specific for timing set to on
/*
    float *time_per_iteration, *time_supp_set, *cg_per_iteration, *time_for_cg;
    if (timingFlag == 1){
      time_per_iteration = (float*)malloc( sizeof(float) * maxiter );
      SAFEmalloc_float(time_per_iteration, "time_per_iteration");
      time_supp_set = (float*)malloc( sizeof(float) * maxiter );
      SAFEmalloc_float(time_supp_set, "time_supp_set");
      if (strcmp(algstr, "HTP")==0){  // CG info only output for HTP
        cg_per_iteration = (float*)malloc( sizeof(float) * maxiter );
	SAFEmalloc_float(cg_per_iteration, "cg_per_iteration");
	time_for_cg = (float*)malloc( sizeof(float) * maxiter );
	SAFEmalloc_float(time_for_cg, "time_for_cg");
      }
    }

*/

/*
    #ifdef VERBOSE
    if (nlhs == 3) { printf("VERBOSE inactive when passing a problem directly.");}
      float *h_Mat_input, *h_Mat, *h_y, *h_Grad, *h_Grad_proj;
      float *h_U, *h_S;
    if (nlhs == 6) {
      h_Mat_input = (float*)malloc( sizeof(float) * mn );
      h_Mat = (float*)malloc( sizeof(float) * mn );
      h_Grad_proj = (float*)malloc( sizeof(float) * mn );
      h_Grad = (float*)malloc( sizeof(float) * mn );
      h_y = (float*)malloc( sizeof(float) * p );
      h_U = (float*)malloc( sizeof(float) * mr );
      h_S = (float*)malloc( sizeof(float) * r );
    }
    #endif
*/


/*
*****************************************
** Set the kernel execution parameters:                                                  
** For device flexibility, find max threads for current device.                         
** Then use this to determine the different kernel execution configurations you may need.
*****************************************
*/


    dim3 threadsPerBlockmn(threads_perblockmn);
    int num_blocksmn = (int)ceil((float)mn/(float)threads_perblockmn);
    dim3 numBlocksmn(num_blocksmn);

    dim3 threadsPerBlockm(threads_perblockm);
    int num_blocksm = (int)ceil((float)m/(float)threads_perblockm);
    dim3 numBlocksm(num_blocksm);

    dim3 threadsPerBlockp(threads_perblockp);
    int num_blocksp = (int)ceil((float)p/(float)threads_perblockp);
    dim3 numBlocksp(num_blocksp);

    dim3 threadsPerBlocknr(threads_perblocknr);
    int num_blocksnr = (int)ceil((float)nr/(float)threads_perblocknr);
    dim3 numBlocksnr(num_blocksnr);

		dim3 threadsPerBlockmr(threads_perblockmr);
    int num_blocksmr = (int)ceil((float)mr/(float)threads_perblockmr);
    dim3 numBlocksmr(num_blocksmr);
		//added by Chenshuai Sui

// initialize cublas library
    cublasInit();
		hipblasHandle_t handle;
		hipblasCreate(&handle);
    SAFEcublas("cublasInit in gagamc_entry.");

/*
*****************************
** CREATE A RANDOM PROBLEM **
*****************************
*/


    if (nlhs == 7){
      createProblem_entry(d_Mat, h_Mat_input, d_U, d_V, d_Y, d_y, d_A, m, n, r, p, matrixEnsemble, &seed, gen, threadsPerBlockp, numBlocksp, threadsPerBlockmn, numBlocksmn, handle);
			curandCheck = hiprandGenerateNormal(gen, d_U, mr, 0,1);
			SAFEcurand(curandCheck, "hiprandGenerateNormal d_U");
			curandCheck = hiprandGenerateNormal(gen, d_V, nr, 0,1);
			SAFEcurand(curandCheck, "hiprandGenerateNormal d_V");

    }
    else if (nlhs == 3){
      hipMemcpy(d_Y, h_Y, sizeof(float)*mn, hipMemcpyHostToDevice);
      hipMemcpy(d_y, h_y, sizeof(float)*p, hipMemcpyHostToDevice);
      hipMemcpy(d_A, h_A_in, sizeof(int)*p, hipMemcpyHostToDevice);
			curandCheck = hiprandGenerateNormal(gen, d_U, mr, 0,1);
			SAFEcurand(curandCheck, "hiprandGenerateNormal d_U");
			curandCheck = hiprandGenerateNormal(gen, d_V, nr, 0,1);
			SAFEcurand(curandCheck, "hiprandGenerateNormal d_V");


    }
    

//  ensure initial approximation is the zero vector
    zero_vector_float<<< numBlocksmn, threadsPerBlockmn >>>((float*)d_Mat, mn);
    SAFEcuda("zero_vector_float in random problem in gagamc_entry");

/*
    #ifdef VERBOSE
    if (nlhs == 6) {
    if (verb>3) {  printf("After createProblem, k = %d \n", k);}
    if (verb>0) {

    printf("The problem size is (m, n, r, p) = (%d, %d, %d, %d).\n", m, n, r, p);

    hipMemcpy(h_A, d_A, sizeof(float)*mn, hipMemcpyDeviceToHost);
    hipMemcpy(h_vec_input, d_vec_input, sizeof(float)*n, hipMemcpyDeviceToHost);
    hipMemcpy(h_vec, d_vec, sizeof(float)*n, hipMemcpyDeviceToHost);
    hipMemcpy(h_y, d_y, sizeof(float)*m, hipMemcpyDeviceToHost);

    printf("Create Problem has provided the following:\n");
    printf("The matrix entries:\n");
    for (int jj=0; jj<min(mn,q); jj++){ printf("%f\n ", h_A[jj]); }
    printf("The initial target vector (x):\n");
    for (int jj=0; jj<min(n,q); jj++){ printf("%f\n", h_vec_input[jj]); }
    printf("The input measurements:\n");
    for (int jj=0; jj<min(m,q); jj++){ printf("%f\n", h_y[jj]); }
    printf("The initial approximation:\n");
    for (int jj=0; jj<min(n,q); jj++){ printf("%f\n", h_vec[jj]); }
    }
    }
    #endif
*/


/*
*************************************************
** Solve this problem with the input algorithm **
*************************************************
*/


    hipEvent_t startALG, stopALG;
    float timeALG;
    hipEventCreate(&startALG);
    hipEventCreate(&stopALG);
    hipEventRecord(startALG,0);


// Initialization of parameters and cublas

    int   iter  = 0;
//    float err   = 0;
//    int   sum   = 0;

    float time_sum=0.0f;

    if (strcmp(algstr, "NIHT")==0) alg = 0;
    else if (strcmp(algstr, "SVT")==0) alg = 1;
    else if (strcmp(algstr, "CGIHT")==0) alg = 2;
		else if (strcmp(algstr, "LMaFit")==0) alg = 3;

    switch (alg) {
	case 0:
	   NIHT_MC_S_entry_SPI(d_Mat, Grad, Grad_proj, d_Y, d_U, d_S, d_V, d_A, d_y, d_MM, d_U_prev, d_R, h_S, h_S_prev, residNorm_prev, m, n, r, p, mn, maxiter, tol, PSVDmaxiter, PSVDtol, gen, &iter, &time_sum, threadsPerBlockp, numBlocksp, threadsPerBlocknr, numBlocksnr, threadsPerBlockmn, numBlocksmn, handle);
	   SAFEcuda("NIHT_MC_S_entry_SPI in gagamc_entry_suic");
	   break; 
	case 1:
	   SVT_MC_S_entry_SPI(d_Mat, Grad, Grad_proj, d_Y, d_U, d_S, d_V, d_A, d_MM, d_U_prev, d_U_init, d_R, h_S, h_S_prev, residNorm_prev, m, n, p_r, p, mn, maxiter, tol, PSVDmaxiter, PSVDtol, &iter, &time_sum, threadsPerBlockp, numBlocksp,threadsPerBlockmn, numBlocksmn,max_threads_per_block, handle);
  	   SAFEcuda("SVT_MC_S_entry_SPI in gagamc_entry");
           break;
	case 2:
	   CGIHT_MC_S_entry_SPI(d_Mat, Grad, Grad_proj, Grad_prev, Grad_prev_proj, d_Y, d_U, d_S, d_V, d_A, d_y, d_y_work, d_MM, d_U_prev, d_R, h_S, h_S_prev, residNorm_prev, m, n, r, p, mn, maxiter, tol, PSVDmaxiter, PSVDtol, gen, &iter, &time_sum, threadsPerBlockp, numBlocksp, threadsPerBlocknr, numBlocksnr, threadsPerBlockmn, numBlocksmn, handle);
	   SAFEcuda("CGIHT_MC_S_entry_SPI in gagamc_entry");
	   break; 

	case 3:
	   LMaFit_MC_S_entry(d_Mat, Grad, d_Y, d_U, d_V, d_A, d_RR, d_U_prev, d_V_prev, Grad_proj, d_SS,d_R, residNorm_prev, m, n, r, p, mn, mr, nr, maxiter, tol, gen, &iter, &time_sum, threadsPerBlockp, numBlocksp, threadsPerBlockmr, numBlocksmr,threadsPerBlocknr, numBlocksnr,  threadsPerBlockmn, numBlocksmn, handle);
	   SAFEcuda("LMaFit_MC_S_entry in gagamc_entry");
	   break; 

	default:
	   printf("[gagamc_entry] Error: The possible (case sensitive) input strings for algorithms using gagamc_entry are:\n NIHT\n SVT\n CGIHT\n LMaFit \n");
	   break;
    }


  
    hipDeviceSynchronize();
		SAFEcuda("hipDeviceSynchronize");
    hipEventRecord(stopALG,0);
		SAFEcuda("hipEventRecord");
    hipEventSynchronize(stopALG);
		SAFEcuda("hipDeviceSynchronize: stopALG");
    hipEventElapsedTime(&timeALG, startALG, stopALG);
		SAFEcuda("hipEventElapsedTime in gagamc_entry_suic:");
    hipEventDestroy(startALG);
		SAFEcuda("hipEventDestroy in gagamc_entry_suic: startALG");
    hipEventDestroy(stopALG);
		SAFEcuda("hipEventDestroy in gagamc_entry_suic: stopALG");




/*
***********************
** Check the Results **
***********************
*/

 /*   if ( (strcmp(algstr, "CGIHT")==0) && (restartFlag==1) ) {
      strcat(algstr,"restarted");
    }
*/
// some CPU action is needed before the results

    if (nlhs == 7){

      results_entry(d_Mat, Grad, h_Mat_input, h_Mat_out, residNorm_prev, h_norms, h_times, convergence_rate, total_iter, d_A, h_A_out, iter, timeALG, time_sum, m, n, r, p, mn, convRateNum, seed, p_startTest, p_stopTest, algstr, handle);
			SAFEcuda("results_entry in gagamc_entry_suic");

/*
      hipMemcpy(h_Mat_out, d_Mat, sizeof(float)*mn, hipMemcpyDeviceToHost);
      hipMemcpy(h_A_out, d_A, sizeof(int)*p, hipMemcpyDeviceToHost);

      total_iter[0] = iter;

      float convRate, root;
      int temp = min(iter, convRateNum);
      root = 1/(float)temp;
      temp=convRateNum-temp;
      convRate = (residNorm_prev[convRateNum-1]/residNorm_prev[temp]);
      convRate = pow(convRate, root);

      convergence_rate[0]=convRate;


      // formulate the norms by computing the matrix Mat_input - Mat which is stored here in Grad
      hipMemcpy(Grad, h_Mat_input, mn*sizeof(float), hipMemcpyHostToDevice);
      float norm2_mat_in = hipblasSnrm2(mn, Grad, 1); // l2 norm of original matrix
      float norm1_mat_in = hipblasSasum(mn, Grad, 1); // l1 norm of original matrix
      hipblasSaxpy(mn, -1.0, d_Mat, 1, Grad, 1);
      h_norms[0]=hipblasSnrm2(mn, Grad, 1)/norm2_mat_in; // l2 norm
      h_norms[1]=hipblasSasum(mn, Grad, 1)/norm1_mat_in; // l1 norm
      // use iter and root as locations for temporary storage in order to compute the l_infity norm    
      iter = hipblasIsamax(mn, Grad, 1);
      hipMemcpy(&root, Grad+iter, sizeof(float), hipMemcpyDeviceToHost);
      h_norms[2]=abs(root);  // l infinity norm


      //  record the timings
      float timeTest;
      h_times[1] = timeALG;
      h_times[2] = time_sum/(float)iter;
      hipDeviceSynchronize();
      hipEventRecord(stopTest,0);
      hipEventSynchronize(stopTest);
      hipEventElapsedTime(&timeTest, startTest, stopTest);
      hipEventDestroy(startTest);
      hipEventDestroy(stopTest);
      h_times[0] = timeTest;
*/
    }  // closes if (nlhs == 7)
    else if (nlhs == 3){
      hipMemcpy(h_Mat_out, d_Mat, sizeof(float)*mn, hipMemcpyDeviceToHost);

      total_iter[0] = iter;

      float convRate, root;
      int temp = min(iter, convRateNum);
      root = 1/(float)temp;
      temp=convRateNum-temp;
      convRate = (residNorm_prev[convRateNum-1]/residNorm_prev[temp]);
      convRate = pow(convRate, root);

      convergence_rate[0]=convRate;
    }  // closes else if (nlhs ==3)
/*
    #ifdef VERBOSE
    if (nlhs == 6){
      printf("Results:\n");
      printf("l2 error = %f\n", h_norms[0]);
      printf("l1 error = %f\n", h_norms[1]);
      printf("l-infinty error = %f\n", h_norms[2]);
      printf("ALG Time = %f ms.\n", h_times[0]);
      printf("Average Iteration Time = %f ms.\n", h_times[1]);
      printf("Total Time (including problem generation) = %f ms.\n", h_times[2]);
      printf("Total iterations = %d\n", iter);
      printf("Support Set identification:\n");
      printf("\t True Positive = %d\n", checkSupport[0]);
      printf("\t False Positive = %d\n", checkSupport[1]);
      printf("\t True Negative = %d\n", checkSupport[2]);
      printf("\t False Negative = %d\n", checkSupport[3]);
      printf("Convergence Rate = %f\n", convergence_rate[0]);
    }
    #endif 
*/

/*
**************
** CLEANUP  **
**************
*/


// free up the allocated memory on the device


    hipFree(d_Mat);
		SAFEcuda("hipFree in gagamc_entry_suic: d_Mat");
    hipFree(Grad);
		SAFEcuda("hipFree in gagamc_entry_suic: Grad");
    hipFree(Grad_proj);
		SAFEcuda("hipFree in gagamc_entry_suic: Grad_proj");
    hipFree(d_Y);
		SAFEcuda("hipFree in gagamc_entry_suic: d_Y");
    hipFree(d_A);
		SAFEcuda("hipFree in gagamc_entry_suic: d_A");
    hipFree(d_U);
		SAFEcuda("hipFree in gagamc_entry_suic: d_U");
    hipFree(d_S);
		SAFEcuda("hipFree in gagamc_entry_suic: d_S");
    hipFree(d_V);
		SAFEcuda("hipFree in gagamc_entry_suic: d_V");
    hipFree(d_y);
		SAFEcuda("hipFree in gagamc_entry_suic: d_y");
    hipFree(d_u);
		SAFEcuda("hipFree in gagamc_entry_suic: d_u");
    hipFree(d_u_prev);
		SAFEcuda("hipFree in gagamc_entry_suic: d_u_prev");
    hipFree(d_v);
		SAFEcuda("hipFree in gagamc_entry_suic: d_v");
    hipFree(d_MM);
		SAFEcuda("hipFree in gagamc_entry_suic: d_MM");
    hipFree(d_U_prev);
		SAFEcuda("hipFree in gagamc_entry_suic: d_U_prev");
    hipFree(d_R);
		SAFEcuda("hipFree in gagamc_entry_suic: d_R");


    if (alg==2){
      hipFree(Grad_prev);
			SAFEcuda("hipFree in gagamc_entry_suic: Grad_prev");
      hipFree(Grad_prev_proj);
			SAFEcuda("hipFree in gagamc_entry_suic: Grad_prev_proj");
      hipFree(d_y_work);
			SAFEcuda("hipFree in gagamc_entry_suic: d_y_work");
    }
		else if(alg==1){
			hipFree(d_U_init);
   		SAFEcuda("hipFree in gagamc_entry_suic: d_U_init");
		}
		else if(alg==3){
			hipFree(d_V_prev);
			SAFEcuda("hipFree in gagamc_entry_suic: d_V_prev");
			hipFree(d_RR);
			SAFEcuda("hipFree in gagamc_entry_suic: d_RR");
			hipFree(d_SS);
			SAFEcuda("hipFree in gagamc_entry_suic: d_SS");
		}
 
	

    curandCheck = hiprandDestroyGenerator(gen);
    SAFEcurand(curandCheck, "hiprandDestroyGenerator in gagamc_entry");

   
		hipblasDestroy(handle);
		cublasShutdown();
    SAFEcublas("cublasShutdown");


// free the memory on the host

    free(residNorm_prev);
    free(h_S);
    free(h_S_prev);
    

/*
    #ifdef VERBOSE
    if (nlhs == 6) {
      free(h_vec_input);
      free(h_vec);
      free(h_vec_thres);
      free(h_grad);
      free(h_y);
      free(h_resid);
      free(h_resid_update);
      free(h_A);
    }
    #endif
*/
  }  //closes the else ensuring the algorithm input was valid

  }  //closes the else ensuring a correct number of input and output arguments

  return;
}



