#include "hip/hip_runtime.h"

/*
**************************************************
**	deterministic-robust-l0			**
**  Noise robust version to parallel_l0		**
**************************************************
*/

inline void deterministic_robust_l0(float *d_vec, float *d_y, float *d_res, int *d_rows, int *d_cols, float *d_vals, int *d_bin, int *d_bin_counters, int *h_bin_counters, const int num_bins, int *p_sum, float tol, const int maxiter, const int k, const int m, const int n, const int d, const int alpha, const int nz, float noise_level, float *resRecord, float *timeRecord, int *p_iter, int debug_mode, dim3 numBlocks, dim3 threadsPerBlock, dim3 numBlocksnp, dim3 threadsPerBlocknp, dim3 numBlocksm, dim3 threadsPerBlockm, dim3 numBlocks_bin, dim3 threadsPerBlock_bin){

	int iter = *p_iter;
	int offset = 0;
	timeRecord[0] = 0.0;

	// Options/Inputs in robust_l0
	int enforce_l1_decrease = 0;
	int do_hard_thresholding = 1; // We always do hard thresholding

	float h_sigma_noise = noise_level;
	float h_sigma_s = 1.0;

	// Thresholding variables
	int k_bin = 0;
	float alpha_ht = 0.25f;
	int MaxBin = (int)(num_bins * (1 - alpha_ht));
	float minVal = 0.0f;
	float maxChange = 1.0f;
	float max_value = MaxMagnitude(d_vec, n);
	float slope = ((num_bins - 1)/(max_value));
	
	float * d_Ax;
	float * d_updates;
	
	hipMalloc((void**)&d_Ax, m * sizeof(float));
	hipMalloc((void**)&d_updates, n * sizeof(float));

	computeResidual(d_res, d_y, d_Ax, d_vec, d_rows, d_cols, d_vals, nz, m, n,
		 numBlocksm, threadsPerBlockm, numBlocksnp, threadsPerBlocknp);
	
	float norm_res;
	float norm_res_start;
	norm_res = hipblasSnrm2(m, d_res, 1);
	norm_res_start = norm_res;
	resRecord[0] = norm_res;


	int isCycling = 0;

	float *resCycling;
	int resCyclingLength = 2*d;
	int resRepeatLength = d - 1;
	resCycling = (float*) malloc(sizeof(float)*resCyclingLength);
	for (int i = 0; i < resCyclingLength; i++)
		resCycling[i] = 0.0;

	float *residNorm_prev;
	float *residNorm_evolution;
	float residNorm_diff = 1.0f;
	int residNorm_length = 2*d;
	residNorm_prev = (float*) malloc(sizeof(float)*residNorm_length);
	residNorm_evolution = (float*) malloc(sizeof(float)*residNorm_length);
	for (int i = 0; i < residNorm_length; i++){
		residNorm_prev[i] = 0.0;
		residNorm_evolution[i] = 1.0f;
	}

        // CUDA constants for p.d.f. construction
	float rho = ((float)k)/((float) m);

	float h_prob_zero_factor = expf(d*rho) - 1;
	float h_prob_equal_factor = expf(2*d*rho) - 1;
	float h_sigma_signal_zero = sqrtf(h_sigma_noise*h_sigma_noise + h_sigma_s*h_sigma_s*d*rho/(1- expf(-d*rho)));
	float h_sigma_signal_equal = sqrtf(2*h_sigma_noise*h_sigma_noise + 2*h_sigma_s*h_sigma_s*d*rho/(1 - expf(-2*d*rho)));

	hipMemcpyToSymbol(HIP_SYMBOL(sigma_noise), &h_sigma_noise, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(prob_zero_factor), &h_prob_zero_factor, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(prob_equal_factor), &h_prob_equal_factor, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sigma_signal_zero), &h_sigma_signal_zero, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sigma_signal_equal), &h_sigma_signal_equal, sizeof(float));

	// Vector or random variables and scores

	float *d_pz_u;
	int *d_scores;

	hipMalloc((void**)&d_pz_u, n*sizeof(float));
	hipMalloc((void**)&d_scores, n*sizeof(int));

	float *d_average_updates;
	hipMalloc((void**)&d_average_updates, n*sizeof(float));

	float prob_thresh = 0.99;

	float norm_res_mean = (h_sigma_noise*h_sigma_noise)*((float) m);
	float norm_res_sd = (h_sigma_noise*h_sigma_noise)*sqrtf(2 * ((float) m)); 

	// Create tmp vectors for residual and signal

	float * d_vec_tmp;
	float * d_res_tmp;

	hipMalloc((void**)&d_vec_tmp, n*sizeof(float));
	hipMalloc((void**)&d_res_tmp, m*sizeof(float));

	hipMemcpy(d_vec_tmp, d_vec, sizeof(float)*n, hipMemcpyDeviceToDevice);
	hipMemcpy(d_res_tmp, d_res, sizeof(float)*m, hipMemcpyDeviceToDevice);

	float norm_res_tmp;
	norm_res_tmp = norm_res;

	int num_failed_attempts = 0;

	while( (iter < maxiter) & (norm_res*norm_res - norm_res_mean > tol*norm_res_sd) & (norm_res < (100*norm_res_start)) & (residNorm_diff > 0.000001) & (isCycling == 0) & (prob_thresh > 0.05) & (num_failed_attempts < d - 1)){

		// time variables
		hipEvent_t start, stop;
		float time;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		// BEGIN STEP
		offset = iter % d;
		
		// compute l0-scores in parallel
		// d_unif has "n" chunks of size "2*d + 1"
		zero_vector_float<<<numBlocks, threadsPerBlock>>>(d_average_updates, n);
		hipDeviceSynchronize();

		// Takes "1" uniform random variable.
		cuda_deterministic_prob_zero_u<<<numBlocks, threadsPerBlock>>>(d_pz_u, d, n, d_res_tmp, d_rows, offset);
		hipDeviceSynchronize();

		// Compute scores
		cuda_compute_scores_det_robust_l0<<<numBlocks, threadsPerBlock>>>(d_scores, d_pz_u, d_average_updates, d, n, d_res_tmp, d_rows, offset, prob_thresh);
		hipDeviceSynchronize();

		// Compute update signal robust l0
		cuda_update_signal_det_robust_l0<<<numBlocks, threadsPerBlock>>>(d_res_tmp, d_vec_tmp, d_updates, d_average_updates, d_rows, n, d, alpha, d_scores, enforce_l1_decrease); 
		hipDeviceSynchronize();

		// compute residual
		computeResidual(d_res_tmp, d_y, d_Ax, d_vec_tmp, d_rows, d_cols, d_vals, nz, m, n,
			 numBlocksm, threadsPerBlockm, numBlocksnp, threadsPerBlocknp);
		hipDeviceSynchronize();

		// hard-thresholding

		if (do_hard_thresholding == 1){
			H_k(d_vec_tmp, k, n, d_bin, d_bin_counters, h_bin_counters, &maxChange, &max_value,
				&slope, &minVal, &alpha_ht, &MaxBin, &k_bin, p_sum, num_bins,
				numBlocks, threadsPerBlock, numBlocks_bin, threadsPerBlock_bin);

			// recompute residual
			computeResidual(d_res_tmp, d_y, d_Ax, d_vec_tmp, d_rows, d_cols, d_vals, nz, m, n, 
				numBlocksm, threadsPerBlockm, numBlocksnp, threadsPerBlocknp);
		}

		norm_res_tmp = hipblasSnrm2(m, d_res_tmp, 1);

		// END STEP

		if (norm_res_tmp < 0.99*norm_res){
			// continue

			hipMemcpy(d_vec, d_vec_tmp, sizeof(float)*n, hipMemcpyDeviceToDevice);
			hipMemcpy(d_res, d_res_tmp, sizeof(float)*m, hipMemcpyDeviceToDevice);
			norm_res = norm_res_tmp;

			// check for no change in residual 
			for (int i = 0; i < residNorm_length - 1; i++){
				residNorm_prev[i] = residNorm_prev[i + 1];
			}
			residNorm_prev[residNorm_length - 1] = norm_res;
			for (int i = 0; i < residNorm_length - 1; i++){
				residNorm_evolution[i] = residNorm_evolution[i + 1];
			}
			residNorm_evolution[residNorm_length - 1] = residNorm_prev[residNorm_length - 2] - residNorm_prev[residNorm_length - 1];
			residNorm_diff = max_list(residNorm_evolution, residNorm_length);

			// Check for cycling
			// Check works, but not in all cases. Need more precise check.
			for (int i = 0; i < resCyclingLength - 1; i++)
				resCycling[i] = resCycling[i + 1]; 
			resCycling[resCyclingLength - 1] = norm_res;
			if (iter > resCyclingLength){
				isCycling = 1;
				for (int i = 3; i < resCyclingLength; i = i + 2)
					isCycling = isCycling*(resCycling[i] == resCycling[i - 2]);
			}
			if (iter > resRepeatLength){
				isCycling = 1;
				for (int i = resCyclingLength - 1; i >= resCyclingLength - resRepeatLength; i = i - 1)
					isCycling = isCycling*(abs(resCycling[i] - resCycling[i-1]) <= 1e-10);

			}

			num_failed_attempts = 0;

		}else{

			hipMemcpy(d_vec_tmp, d_vec, sizeof(float)*n, hipMemcpyDeviceToDevice);
			hipMemcpy(d_res_tmp, d_res, sizeof(float)*m, hipMemcpyDeviceToDevice);

			prob_thresh -= 0.05;
			num_failed_attempts += 1;
		}

		// end timing
		hipDeviceSynchronize();
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);
	
		iter = iter + 1;
		resRecord[iter] = norm_res;
		timeRecord[iter] = timeRecord[iter - 1] + time;

		if(debug_mode == 1){
			printf("iter = %d, norm_res = %5.10f, prob_thresh = %1.2f, isCycling = %d\n", iter, norm_res, prob_thresh, isCycling);
		}


	}

	float norm_xhat = 0.0;
	float big_float = 10000.0;
	
	norm_xhat = hipblasSnrm2(n, d_vec, 1);

	// If norm_xhat == 0.0, then no updates were performed.
	// We increase the norm so that GAGA doesn't think it is a success (when n is large)
	if (norm_xhat == 0.0){
		hipMemcpy(d_vec, &big_float, sizeof(float), hipMemcpyHostToDevice);
	}
	

	*p_iter = iter;

	free(resCycling);
	free(residNorm_prev);
	free(residNorm_evolution);

	// clean GPU
	hipFree(d_Ax);
	hipFree(d_updates);
        //hipFree(randArray);
        //hipFree(d_state);

	hipFree(d_average_updates);
	hipFree(d_pz_u);
	hipFree(d_scores);

	// tmp vectors
	hipFree(d_vec_tmp);
	hipFree(d_res_tmp);

}

