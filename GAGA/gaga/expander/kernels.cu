
#include <hip/hip_runtime.h>


__global__ void cuda_update_residual(float *d_res, int *d_rows, float *d_updates, int n, int p){
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if (xIndex < n){
		if (abs(d_updates[xIndex]) > 0){
                	int idx = xIndex*p;
			float update = -d_updates[xIndex]; 
			for (int i = 0; i < p; i++){
				atomicAdd(d_res + d_rows[idx + i], update);
			}
		} 
	}
}

// Not tested yet
/*
__global__ void cuda_update_residual_np(float *d_res, int *d_rows, float *d_updates, int n, int p){
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < n*p){
                int idx = (int) (tid/p);
		float update = -d_updates[idx]; 
		if (abs(d_updates[xIndex]) > 0){
			atomicAdd(d_res + d_rows[idx + i], update);
		}
	}
}
*/

/*
*************************************************
* 	For tranforming to row-major order	*
*************************************************
*/

// nonzero_rows_count[i] = number of nonzeros in row i.
__global__ void count_nonzeros_in_rows_index(float *nonzero_rows_count, int *rm_rows_index, int m){
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if ( xIndex < m){
		nonzero_rows_count[xIndex] = (float)(rm_rows_index[2*xIndex + 1] - rm_rows_index[2*xIndex] + 1);
	}
}

// Create vector 2-m vector [a_1, a_2, b_1, b_2, ..., x_1, x_2] indicating start and end position of columns in row i of matrix
// i.e. if row_i has only one nonzero, then a_i == b_i
__global__ void create_row_index(int *rm_rows, int *rm_rows_index, int m, int np){
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if ( xIndex < np){
		if (xIndex > 0 && xIndex < np - 1){
			if (rm_rows[xIndex - 1] < rm_rows[xIndex]){
				rm_rows_index[2*rm_rows[xIndex]] = xIndex;
			}
			if(rm_rows[xIndex] < rm_rows[xIndex + 1]){
				rm_rows_index[2*rm_rows[xIndex] + 1] = xIndex;
			}
		}
		if (xIndex == 0){
			rm_rows_index[2*rm_rows[xIndex]] = xIndex;
			if(rm_rows[xIndex] < rm_rows[xIndex + 1])
				rm_rows_index[2*rm_rows[xIndex] + 1] = xIndex;
		}
		if (xIndex == np - 1){
			if (rm_rows[xIndex - 1] < rm_rows[xIndex])
				rm_rows_index[2*rm_rows[xIndex]] = xIndex;
			rm_rows_index[2*rm_rows[xIndex] + 1] = xIndex;
		}
	}
}


/*
*************************************************
*						*
* 			SMP			*
*						*
*************************************************
*/

__global__ void cuda_compute_scores_smp(float *d_u, float *d_b, int *d_rows, float *d_aux, int n, int p)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	if(tid < n){
		int idx = p*tid;
		float tmp = 0.0;
		float median = 0.0;

		for(int i = 0; i < p; i++)
			d_aux[idx + i] = (float) d_b[d_rows[idx + i]];
	
		// sort chunk in d_aux with bubblesort. O(p^2) work for a thread, but O(1) memory 
		// We exploit computational power

		for(int i = 0; i < p; i++)
			for(int j = 0; j < p - 1; j++)
				if(d_aux[idx + j] > d_aux[idx + j + 1]){
					tmp = d_aux[idx + j + 1];
					d_aux[idx + j + 1] = d_aux[idx + j];
					d_aux[idx + j] = tmp;
				}	

		median = (p % 2) == 0 ? (d_aux[idx + (p/2) - 1] + d_aux[idx + (p/2)])/2 : d_aux[idx + (p - 1)/2];	
		d_u[tid] = median;
	}
}


/*
*************************************************
*						*
* 			SSMP			*
*						*
*************************************************
*/

__global__ void cuda_compute_scores_ssmp(float *d_scores, float *d_medians, float *d_res, int *d_rows, float *d_vals, float *d_aux, int n, int p)
{
        int tid = threadIdx.x + blockDim.x*blockIdx.x;
        if(tid < n){

                int idx = p*tid;
                float tmp = 0.0;
                float z = 0.0;

                for(int i = 0; i < p; i++)
                        d_aux[idx + i] = d_res[d_rows[idx + i]];

                // sort chunk in d_aux with bubblesort. O(p^2) work for a thread, but O(1) memory 
                // We exploit computational power

                for(int i = 0; i < p; i++)
                        for(int j = 0; j < p - 1; j++)
                                if(d_aux[idx + j] > d_aux[idx + j + 1]){
                                        tmp = d_aux[idx + j + 1];
                                        d_aux[idx + j + 1] = d_aux[idx + j];
                                        d_aux[idx + j] = tmp;
                                }

                // z = median
                z = (p % 2) == 0 ? (d_aux[idx + (p/2) - 1] + d_aux[idx + (p/2)])/2 : d_aux[idx + (p - 1)/2];

		d_medians[tid] = z;
		d_scores[tid] = abs(z);

        }
}

__global__ void cuda_update_scores_ssmp(float *d_scores, float *d_medians, float *d_res, int *d_rows, float *d_aux, int *d_rm_cols, int *d_rm_rows_index, int max_nonzero_rows_count, int p, int h_node)
{
        int tid = threadIdx.x + blockDim.x*blockIdx.x;
        if(tid < max_nonzero_rows_count*p){
		int section = (int)(tid/max_nonzero_rows_count); // from 0 to p-1
		int row = d_rows[p*h_node + section];
		int num_nonzeros_in_row = d_rm_rows_index[2*row + 1] - d_rm_rows_index[2*row] + 1; // from 1 to max_nonzero_rows_count
		int max_position = num_nonzeros_in_row - 1; // from 0 to (max_nonzero_rows_count - 1)
		int position_in_section = tid - section*max_nonzero_rows_count; // from 0 to (max_nonzero_rows_count - 1) 
		if (position_in_section <= max_position){

			int node = d_rm_cols[d_rm_rows_index[2*row] + position_in_section];
			int row_idx = p*node;
			int idx = p*tid;
			float tmp = 0.0;
			float z = 0.0;

			for(int i = 0; i < p; i++)
				d_aux[idx + i] = d_res[d_rows[row_idx + i]];

			// sort chunk in d_aux with bubblesort. O(p^2) work for a thread, but O(1) memory 

			for(int i = 0; i < p; i++)
				for(int j = 0; j < p - 1; j++)
					if(d_aux[idx + j] > d_aux[idx + j + 1]){
						tmp = d_aux[idx + j + 1];
						d_aux[idx + j + 1] = d_aux[idx + j];
						d_aux[idx + j] = tmp;
					}

			// z = median
			z = (p % 2) == 0 ? (d_aux[idx + (p/2) - 1] + d_aux[idx + (p/2)])/2 : d_aux[idx + (p - 1)/2];

			d_medians[node] = z;
			d_scores[node] = abs(z);
		}
        }
}


/*
*************************************************
*						*
* 			ER			*
*						*
*************************************************
*/

__global__ void cuda_compute_scores_er(float *d_scores, float *d_modes, float *d_res, int *d_rows, float *d_vals, float *d_aux, int n, int p)
{
        int tid = threadIdx.x + blockDim.x*blockIdx.x;
        if(tid < n){

                int idx = p*tid;
                float tmp = 0.0;
		float EPS = 0.0000001;

                for(int i = 0; i < p; i++)
                        d_aux[idx + i] = d_res[d_rows[idx + i]];

                // sort chunk in d_aux with bubblesort. O(p^2) work for a thread, but O(1) memory 
                // We exploit computational power

                for(int i = 0; i < p; i++)
                        for(int j = 0; j < p - 1; j++)
                                if(d_aux[idx + j] > d_aux[idx + j + 1]){
                                        tmp = d_aux[idx + j + 1];
                                        d_aux[idx + j + 1] = d_aux[idx + j];
                                        d_aux[idx + j] = tmp;
                                }

		// Compute modes

		float mode = d_aux[idx];
		float count = 1;
		float countMode = 1;

		for (int i = 1; i < p; i++){
			if (abs(d_aux[idx + i] - d_aux[idx + i - 1]) <= EPS){
				count = count + 1;
				if (count > countMode){
					countMode = count;
					mode = d_aux[idx + i - 1];
				}
			}else{
				count = 1;
			}
		}

		d_modes[tid] = mode;
		d_scores[tid] = abs(mode) > EPS ? countMode : 0;

        }
}

/*
0 <= tid < max_nonzero_rows_count*p
[section_0 | section_1 | ... | section_{p-1}]

*/
__global__ void cuda_update_scores_er(float *d_scores, float *d_modes, float *d_res, int *d_rows, float *d_aux, int *d_rm_cols, int *d_rm_rows_index, int max_nonzero_rows_count, int p, int h_node)
{
        int tid = threadIdx.x + blockDim.x*blockIdx.x;
        if(tid < max_nonzero_rows_count*p){
		int section = (int)(tid/max_nonzero_rows_count); // from 0 to p-1
		int row = d_rows[p*h_node + section];
		int num_nonzeros_in_row = d_rm_rows_index[2*row + 1] - d_rm_rows_index[2*row] + 1; // from 1 to max_nonzero_rows_count
		int max_position = num_nonzeros_in_row - 1; // from 0 to (max_nonzero_rows_count - 1)
		int position_in_section = tid - section*max_nonzero_rows_count; // from 0 to (max_nonzero_rows_count - 1) 
		if (position_in_section <= max_position){

			int node = d_rm_cols[d_rm_rows_index[2*row] + position_in_section];
			int row_idx = p*node;
			int idx = p*tid;
			float tmp = 0.0;
			float EPS = 0.0000001;

			for(int i = 0; i < p; i++)
				d_aux[idx + i] = d_res[d_rows[row_idx + i]];

			// sort chunk in d_aux with bubblesort. O(p^2) work for a thread, but O(1) memory 

			for(int i = 0; i < p; i++)
				for(int j = 0; j < p - 1; j++)
					if(d_aux[idx + j] > d_aux[idx + j + 1]){
						tmp = d_aux[idx + j + 1];
						d_aux[idx + j + 1] = d_aux[idx + j];
						d_aux[idx + j] = tmp;
					}

			// Compute modes
			float mode = d_aux[idx];
			float count = 1;
			float countMode = 1;

			for (int i = 1; i < p; i++){
				if (abs(d_aux[idx + i] - d_aux[idx + i - 1]) <= EPS){
					count = count + 1;
					if (count > countMode){
						countMode = count;
						mode = d_aux[idx + i - 1];
					}
				}else{
					count = 1;
				}
			}

			d_modes[node] = mode;
			d_scores[node] = abs(mode) > EPS ? countMode : 0;
		}
        }
}


/*
*************************************************
*						*
* 		Parallel-LDDSR			*
*						*
*************************************************
*/

__global__ void cuda_compute_scores_lddsr(float *d_scores, float *d_res, float *d_vec, float *d_updates, int *d_rows, int *d_foundCandidates, int n, int d, int shift)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	
	if (tid < n){
		float omega = 0.0;
		int idx = d*tid;
		float EPS = 0.000001;
		float thresh = ((float) d)/2;

		d_scores[tid] = 0;
		omega = d_res[d_rows[idx + shift]];
		d_updates[tid] = 0;

		if (abs(omega) > EPS){
			for (int i = 0; i < d; i++){
				if (abs(d_res[d_rows[idx + i]] - omega) <= EPS)
					d_scores[tid]++;
			}
			if (d_scores[tid] > thresh){
				d_foundCandidates[0] = 1;
				d_vec[tid] = d_vec[tid] + omega; 
				/*
				// update residual
				for (int i = 0; i < d; i++){
					atomicAdd(d_res + d_rows[idx + i], -omega);
				}
				*/
				d_updates[tid] = omega;
			}
			//d_updates[tid] = d_vec[tid] + omega; // Use this when using h_found_candidates solution
		}

	}
}


/*
*************************************************
*						*
* 		Parallel-L0			*
*						*
*************************************************
*/

__global__ void cuda_compute_scores_l0(float *d_scores, float *d_res, float *d_vec, float *d_updates, int *d_rows, int *d_foundCandidates, int n, int d, int thresh, int shift)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	
	if (tid < n){
		float omega = 0.0;
		int idx = d*tid;
		float EPS = 0.000001;
                //float thresh = 1;

		d_scores[tid] = 0;
		omega = d_res[d_rows[idx + shift]];
		d_updates[tid] = 0;

		if (abs(omega) > EPS){
			for (int i = 0; i < d; i++){
				if (abs(d_res[d_rows[idx + i]] - omega) <= EPS)
					d_scores[tid]++;
				if (abs(d_res[d_rows[idx + i]]) <= EPS)
					d_scores[tid]--;
			}
			if (d_scores[tid] > thresh){
				d_foundCandidates[0] = 1;
				d_vec[tid] = d_vec[tid] + omega; 
				/*
				// update residual
				for (int i = 0; i < d; i++){
					atomicAdd(d_res + d_rows[idx + i], -omega);
				}
				*/
				d_updates[tid] = omega;
				//d_updates[tid] = d_vec[tid] + omega;
			}
		}
	}
}

/*
*************************************************
*						*
* For exact thresholding and residual update	*
*						*
*************************************************
*/

__global__ void thresholdK(float *d_vec, int *d_bin, int k_bin, int * offset, int n){
        unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

        if( (xIndex < n) & (d_bin[xIndex] == k_bin) ){
                if(atomicSub(offset, 1) - 1 >= 0){
                        d_vec[xIndex] = 0;
                }
        }
}

__global__ void cudaUpdateSignal(float *d_vec, int h_node, float *d_updates){

        int tid = threadIdx.x + blockDim.x*blockIdx.x;

        if(tid == h_node){
                d_vec[tid] += d_updates[h_node];
        }
}

__global__ void cudaUpdateResidual(float *d_res, int *d_rows, float *d_vals, float *d_medians, int h_node, int p){

        int tid = threadIdx.x + blockDim.x*blockIdx.x;

        if(tid == h_node){
                int idx = tid*p;
                for(int i = 0; i < p; i++)
                        d_res[d_rows[idx + i]] -= d_medians[tid]*(d_vals[idx + i]);
        }
}

__global__ void countSupport( float *d_vec, int n, int *dev_c ) { 
 
        int tid = threadIdx.x + blockIdx.x * blockDim.x; 
        if( tid < n ) { 
                if(d_vec[tid] != 0){ 
                        atomicAdd( dev_c , 1 ); 
                } 
        } 
} 
